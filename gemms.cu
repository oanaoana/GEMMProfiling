#include "hip/hip_runtime.h"
#include "gemms.cuh"
#include <hip/hip_runtime.h>
#include <hipblas.h>  // Add this include
#include <stdio.h>

__global__ void matmul_naive( float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k)
            sum += A[row * N + k] * B[k * N + col];
        C[row * N + col] = sum;}
}

__global__ void matmul_tiled(float *A, float *B, float *C, int N) {
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < N && t * TILE_SIZE + threadIdx.x < N)
            tile_A[threadIdx.y][threadIdx.x] = A[row * N + (t * TILE_SIZE + threadIdx.x)];
        else
            tile_A[threadIdx.y][threadIdx.x] = 0.0f;

        if (t * TILE_SIZE + threadIdx.y < N && col < N)
            tile_B[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            tile_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];

        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}

void launch_cublas(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Setup alpha and beta for sgemm
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Call cuBLAS sgemm
    // Note: cuBLAS uses column-major order while our code uses row-major order
    // So we compute C = B*A as a workaround for row-major C = A*B
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_B, n,  // B matrix
                d_A, n,  // A matrix
                &beta,
                d_C, n); // C matrix

    // Destroy handle
    hipblasDestroy(handle);
}
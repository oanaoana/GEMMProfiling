#include "hip/hip_runtime.h"
// gemms.cu
#include "gemms.cuh"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include "cutlass/gemm/device/gemm.h"

// Naive implementation
__global__ void matmul_naive(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Launch wrapper for naive implementation
void launch_naive(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    matmul_naive<<<blocks, threads>>>(d_A, d_B, d_C, n);
}

// Tiled implementation
__global__ void matmul_tiled(float *A, float *B, float *C, int N) {
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE + 1];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;
    int num_tiles = (N + TILE_SIZE - 1) / TILE_SIZE;

    for (int t = 0; t < num_tiles; ++t) {
        // Load A tile (this was correct)
        int A_col = t * TILE_SIZE + threadIdx.x;
        tile_A[threadIdx.y][threadIdx.x] = (row < N && A_col < N) ?
                                          A[row * N + A_col] : 0.0f;

        // FIXED: Load B tile correctly
        int B_row = t * TILE_SIZE + threadIdx.y;  // Use threadIdx.y for row
        int B_col = col;  // Use the actual column this thread is computing
        tile_B[threadIdx.y][threadIdx.x] = (B_row < N && B_col < N) ?
                                          B[B_row * N + B_col] : 0.0f;

        __syncthreads();

        // Compute partial result
        for (int k = 0; k < TILE_SIZE; ++k)
            sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];

        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}

// Launch wrapper for tiled implementation
void launch_tiled(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    matmul_tiled<<<blocks, threads>>>(d_A, d_B, d_C, n);
}

// cuBLAS implementation
void launch_cublas(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Setup alpha and beta for sgemm
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Call cuBLAS sgemm
    // Note: cuBLAS uses column-major order while our code uses row-major order
    // So we compute C = B*A as a workaround for row-major C = A*B
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_B, n,  // B matrix
                d_A, n,  // A matrix
                &beta,
                d_C, n); // C matrix

    // Destroy handle
    hipblasDestroy(handle);
}

// Add TensorCore implementation
void launch_cublas_tensor(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Enable TensorCore math if supported
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_B, n,
                d_A, n,
                &beta,
                d_C, n);

    hipblasDestroy(handle);
}

// CUTLASS implementation
void launch_cutlass(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Define CUTLASS GEMM kernel
    using Gemm = cutlass::gemm::device::Gemm<
        float,                           // Data type of A matrix
        cutlass::layout::RowMajor,       // Layout of A matrix
        float,                           // Data type of B matrix
        cutlass::layout::RowMajor,       // Layout of B matrix
        float,                           // Data type of C matrix
        cutlass::layout::RowMajor,       // Layout of C matrix
        float                            // Data type for internal accumulation
    >;

    // CUTLASS GEMM arguments
    Gemm::Arguments arguments{
        {n, n, n},          // Problem size (M, N, K)
        {d_A, n},           // Tensor A (ptr, leading dimension)
        {d_B, n},           // Tensor B (ptr, leading dimension)
        {d_C, n},           // Tensor C (ptr, leading dimension)
        {d_C, n},           // Tensor D (ptr, leading dimension) - output
        {1.0f, 0.0f}        // Scalars alpha, beta
    };

    // Initialize CUTLASS GEMM
    Gemm gemm_op;

    // Check if arguments are valid
    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
        printf("CUTLASS GEMM cannot implement these arguments\n");
        return;
    }

    // Initialize the GEMM operator
    status = gemm_op.initialize(arguments);
    if (status != cutlass::Status::kSuccess) {
        printf("Failed to initialize CUTLASS GEMM\n");
        return;
    }

    // Launch the GEMM kernel
    status = gemm_op();
    if (status != cutlass::Status::kSuccess) {
        printf("CUTLASS GEMM kernel launch failed\n");
        return;
    }

    // Synchronize
    hipDeviceSynchronize();
}

// Optional: Advanced CUTLASS with Tensor Cores
void launch_cutlass_tensor(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Using mixed precision for Tensor Cores (requires input conversion)
    using Gemm = cutlass::gemm::device::Gemm<
        cutlass::half_t,                 // Data type of A matrix (FP16)
        cutlass::layout::RowMajor,       // Layout of A matrix
        cutlass::half_t,                 // Data type of B matrix (FP16)
        cutlass::layout::RowMajor,       // Layout of B matrix
        float,                           // Data type of C matrix (FP32)
        cutlass::layout::RowMajor,       // Layout of C matrix
        float,                           // Data type for internal accumulation
        cutlass::arch::OpClassTensorOp,  // Use Tensor Cores
        cutlass::arch::Sm80              // Target SM 8.0+ for RTX 4080
    >;

    // Note: This requires converting FP32 inputs to FP16
    // For simplicity, using regular CUTLASS for now
    // In production, you'd convert d_A and d_B to half precision

    printf("CUTLASS Tensor Core version requires FP16 input conversion\n");
    // Fall back to regular CUTLASS
    launch_cutlass(d_A, d_B, d_C, n, blocks, threads);
}
#include "hip/hip_runtime.h"
// gemms.cu
#include "gemms.cuh"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include "cutlass/gemm/device/gemm.h"

// Naive implementation
__global__ void matmul_naive(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Launch wrapper for naive implementation
void launch_naive(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    matmul_naive<<<blocks, threads>>>(d_A, d_B, d_C, n);
}

// Tiled implementation
__global__ void matmul_tiled(float *A, float *B, float *C, int N) {
    // Shared memory with bank conflict avoidance (+1 padding)
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE + 1];

    // Thread indices
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;

    // Global indices for this thread
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    // Accumulator - keep in register
    float sum = 0.0f;
    int num_tiles = (N + TILE_SIZE - 1) / TILE_SIZE;

    // Main tiling loop
    for (int t = 0; t < num_tiles; ++t) {
        // Load tile A - coalesced memory access
        int A_row = row;
        int A_col = t * TILE_SIZE + tx;
        tile_A[ty][tx] = (A_row < N && A_col < N) ? A[A_row * N + A_col] : 0.0f;

        // Load tile B - coalesced memory access
        int B_row = t * TILE_SIZE + ty;
        int B_col = col;
        tile_B[ty][tx] = (B_row < N && B_col < N) ? B[B_row * N + B_col] : 0.0f;

        // Wait for all threads to finish loading
        __syncthreads();

        // Compute partial dot product with optimizations
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            // Use fused multiply-add for better performance
            sum = __fmaf_rn(tile_A[ty][k], tile_B[k][tx], sum);
        }

        // Wait before loading next tiles
        __syncthreads();
    }

    // Write result to global memory
    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

// Launch wrapper for tiled implementation
void launch_tiled(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    matmul_tiled<<<blocks, threads>>>(d_A, d_B, d_C, n);
}

// cuBLAS implementation
void launch_cublas(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Setup alpha and beta for sgemm
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Call cuBLAS sgemm
    // Note: cuBLAS uses column-major order while our code uses row-major order
    // So we compute C = B*A as a workaround for row-major C = A*B
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_B, n,  // B matrix
                d_A, n,  // A matrix
                &beta,
                d_C, n); // C matrix

    // Destroy handle
    hipblasDestroy(handle);
}

// Add TensorCore implementation
void launch_cublas_tensor(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Enable TensorCore math if supported
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_B, n,
                d_A, n,
                &beta,
                d_C, n);

    hipblasDestroy(handle);
}

// CUTLASS implementation
void launch_cutlass(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Define CUTLASS GEMM kernel
    using Gemm = cutlass::gemm::device::Gemm<
        float,                           // Data type of A matrix
        cutlass::layout::RowMajor,       // Layout of A matrix
        float,                           // Data type of B matrix
        cutlass::layout::RowMajor,       // Layout of B matrix
        float,                           // Data type of C matrix
        cutlass::layout::RowMajor,       // Layout of C matrix
        float                            // Data type for internal accumulation
    >;

    // CUTLASS GEMM arguments
    Gemm::Arguments arguments{
        {n, n, n},          // Problem size (M, N, K)
        {d_A, n},           // Tensor A (ptr, leading dimension)
        {d_B, n},           // Tensor B (ptr, leading dimension)
        {d_C, n},           // Tensor C (ptr, leading dimension)
        {d_C, n},           // Tensor D (ptr, leading dimension) - output
        {1.0f, 0.0f}        // Scalars alpha, beta
    };

    // Initialize CUTLASS GEMM
    Gemm gemm_op;

    // Check if arguments are valid
    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
        printf("CUTLASS GEMM cannot implement these arguments\n");
        return;
    }

    // Initialize the GEMM operator
    status = gemm_op.initialize(arguments);
    if (status != cutlass::Status::kSuccess) {
        printf("Failed to initialize CUTLASS GEMM\n");
        return;
    }

    // Launch the GEMM kernel
    status = gemm_op();
    if (status != cutlass::Status::kSuccess) {
        printf("CUTLASS GEMM kernel launch failed\n");
        return;
    }

    // Synchronize
    hipDeviceSynchronize();
}

// Optional: Advanced CUTLASS with Tensor Cores
void launch_cutlass_tensor(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Using mixed precision for Tensor Cores (requires input conversion)
    using Gemm = cutlass::gemm::device::Gemm<
        cutlass::half_t,                 // Data type of A matrix (FP16)
        cutlass::layout::RowMajor,       // Layout of A matrix
        cutlass::half_t,                 // Data type of B matrix (FP16)
        cutlass::layout::RowMajor,       // Layout of B matrix
        float,                           // Data type of C matrix (FP32)
        cutlass::layout::RowMajor,       // Layout of C matrix
        float,                           // Data type for internal accumulation
        cutlass::arch::OpClassTensorOp,  // Use Tensor Cores
        cutlass::arch::Sm80              // Target SM 8.0+ for RTX 4080
    >;

    // Note: This requires converting FP32 inputs to FP16
    // For simplicity, using regular CUTLASS for now
    // In production, you'd convert d_A and d_B to half precision

    printf("CUTLASS Tensor Core version requires FP16 input conversion\n");
    // Fall back to regular CUTLASS
    launch_cutlass(d_A, d_B, d_C, n, blocks, threads);
}
#include "hip/hip_runtime.h"
// gemms.cu
#include "gemms.cuh"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include "cutlass/gemm/device/gemm.h"

extern int g_pitch_A;  // Declare extern to access from benchmark.cu

// Naive implementation
__global__ void matmul_naive(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Launch wrapper for naive implementation
void launch_naive(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    matmul_naive<<<blocks, threads>>>(d_A, d_B, d_C, n);
}

// Tiled implementation
__global__ void matmul_tiled(float *A, float *B, float *C, int N) {
    // Shared memory with bank conflict avoidance (+1 padding)
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE + 1];

    // Thread indices
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;

    // Global indices for this thread
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    // Accumulator - keep in register
    float sum = 0.0f;
    int num_tiles = (N + TILE_SIZE - 1) / TILE_SIZE;

    // Main tiling loop
    for (int t = 0; t < num_tiles; ++t) {
        // Load tile A - coalesced memory access
        int A_row = row;
        int A_col = t * TILE_SIZE + tx;
        tile_A[ty][tx] = (A_row < N && A_col < N) ? A[A_row * N + A_col] : 0.0f;

        // Load tile B - coalesced memory access
        int B_row = t * TILE_SIZE + ty;
        int B_col = col;
        tile_B[ty][tx] = (B_row < N && B_col < N) ? B[B_row * N + B_col] : 0.0f;

        // Wait for all threads to finish loading
        __syncthreads();

        // Compute partial dot product with optimizations
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            // Use fused multiply-add for better performance
            sum = __fmaf_rn(tile_A[ty][k], tile_B[k][tx], sum);
        }

        // Wait before loading next tiles
        __syncthreads();
    }

    // Write result to global memory
    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

// Launch wrapper for tiled implementation
void launch_tiled(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // For now, assume regular allocation (no pitch)
    matmul_tiled<<<blocks, threads>>>(d_A, d_B, d_C, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
}

// CORRECTED rectangular tiled implementation
__global__ void matmul_tiled_rectangular(float *A, float *B, float *C, int N) {

    __shared__ float tile_A[TILE_M][TILE_K + 1];    // 16×32
    __shared__ float tile_B[TILE_K][TILE_N + 1];    // 32×16

    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_M + ty;
    int col = blockIdx.x * TILE_N + tx;

    float sum = 0.0f;
    int num_tiles = (N + TILE_K - 1) / TILE_K;

    for (int t = 0; t < num_tiles; ++t) {
        // FIXED: Load A tile with proper indexing
        // Each thread loads consecutive elements
        int A_col_base = t * TILE_K;

        // Load first half: threads 0-15 load columns 0-15
        if (tx < TILE_K && row < N && (A_col_base + tx) < N) {
            tile_A[ty][tx] = A[row * N + (A_col_base + tx)];
        } else if (tx < TILE_K) {
            tile_A[ty][tx] = 0.0f;
        }

        // Load second half: threads 0-15 load columns 16-31
        if ((tx + BLOCK_N) < TILE_K && row < N && (A_col_base + tx + BLOCK_N) < N) {
            tile_A[ty][tx + BLOCK_N] = A[row * N + (A_col_base + tx + BLOCK_N)];
        } else if ((tx + BLOCK_N) < TILE_K) {
            tile_A[ty][tx + BLOCK_N] = 0.0f;
        }

        // FIXED: Load B tile with proper indexing
        int B_row_base = t * TILE_K;

        // Load first half: threads row 0-15 load rows 0-15
        if (ty < TILE_K && (B_row_base + ty) < N && col < N) {
            tile_B[ty][tx] = B[(B_row_base + ty) * N + col];
        } else if (ty < TILE_K) {
            tile_B[ty][tx] = 0.0f;
        }

        // Load second half: threads row 0-15 load rows 16-31
        if ((ty + BLOCK_M) < TILE_K && (B_row_base + ty + BLOCK_M) < N && col < N) {
            tile_B[ty + BLOCK_M][tx] = B[(B_row_base + ty + BLOCK_M) * N + col];
        } else if ((ty + BLOCK_M) < TILE_K) {
            tile_B[ty + BLOCK_M][tx] = 0.0f;
        }

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE_K; ++k) {
            sum = __fmaf_rn(tile_A[ty][k], tile_B[k][tx], sum);
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

// Launch wrapper for rectangular tiled implementation
void launch_tiled_rect(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Use TILE_M, TILE_N for block dimensions
    dim3 rect_threads(BLOCK_N, BLOCK_M);
    dim3 rect_blocks((n + TILE_N - 1) / TILE_N, (n + TILE_M - 1) / TILE_M);

    matmul_tiled_rectangular<<<rect_blocks, rect_threads>>>(d_A, d_B, d_C, n);
}

// cuBLAS implementation
void launch_cublas(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Setup alpha and beta for sgemm
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Call cuBLAS sgemm
    // Note: cuBLAS uses column-major order while our code uses row-major order
    // So we compute C = B*A as a workaround for row-major C = A*B
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_B, n,  // B matrix
                d_A, n,  // A matrix
                &beta,
                d_C, n); // C matrix

    // Destroy handle
    hipblasDestroy(handle);
}

// Add TensorCore implementation
void launch_cublas_tensor(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Enable TensorCore math if supported
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_B, n,
                d_A, n,
                &beta,
                d_C, n);

    hipblasDestroy(handle);
}

// CUTLASS implementation
void launch_cutlass(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Define CUTLASS GEMM kernel
    using Gemm = cutlass::gemm::device::Gemm<
        float,                           // Data type of A matrix
        cutlass::layout::RowMajor,       // Layout of A matrix
        float,                           // Data type of B matrix
        cutlass::layout::RowMajor,       // Layout of B matrix
        float,                           // Data type of C matrix
        cutlass::layout::RowMajor,       // Layout of C matrix
        float                            // Data type for internal accumulation
    >;

    // CUTLASS GEMM arguments
    Gemm::Arguments arguments{
        {n, n, n},          // Problem size (M, N, K)
        {d_A, n},           // Tensor A (ptr, leading dimension)
        {d_B, n},           // Tensor B (ptr, leading dimension)
        {d_C, n},           // Tensor C (ptr, leading dimension)
        {d_C, n},           // Tensor D (ptr, leading dimension) - output
        {1.0f, 0.0f}        // Scalars alpha, beta
    };

    // Initialize CUTLASS GEMM
    Gemm gemm_op;

    // Check if arguments are valid
    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
        printf("CUTLASS GEMM cannot implement these arguments\n");
        return;
    }

    // Initialize the GEMM operator
    status = gemm_op.initialize(arguments);
    if (status != cutlass::Status::kSuccess) {
        printf("Failed to initialize CUTLASS GEMM\n");
        return;
    }

    // Launch the GEMM kernel
    status = gemm_op();
    if (status != cutlass::Status::kSuccess) {
        printf("CUTLASS GEMM kernel launch failed\n");
        return;
    }

    // Synchronize
    hipDeviceSynchronize();
}

// Optional: Advanced CUTLASS with Tensor Cores
void launch_cutlass_tensor(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Using mixed precision for Tensor Cores (requires input conversion)
    using Gemm = cutlass::gemm::device::Gemm<
        cutlass::half_t,                 // Data type of A matrix (FP16)
        cutlass::layout::RowMajor,       // Layout of A matrix
        cutlass::half_t,                 // Data type of B matrix (FP16)
        cutlass::layout::RowMajor,       // Layout of B matrix
        float,                           // Data type of C matrix (FP32)
        cutlass::layout::RowMajor,       // Layout of C matrix
        float,                           // Data type for internal accumulation
        cutlass::arch::OpClassTensorOp,  // Use Tensor Cores
        cutlass::arch::Sm80              // Target SM 8.0+ for RTX 4080
    >;

    // Note: This requires converting FP32 inputs to FP16
    // For simplicity, using regular CUTLASS for now
    // In production, you'd convert d_A and d_B to half precision

    printf("CUTLASS Tensor Core version requires FP16 input conversion\n");
    // Fall back to regular CUTLASS
    launch_cutlass(d_A, d_B, d_C, n, blocks, threads);
}

// Add pitched kernel and launch function
__global__ void matmul_tiled_pitch(float *A, float *B, float *C, int N, int pitch_A) {
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE + 1];

    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

    float sum = 0.0f;

    //printf("Debug: Pitched kernel launched with pitch_A = %d\n", pitch_A);
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int A_row = row;
        int A_col = t * TILE_SIZE + tx;
        int B_row = t * TILE_SIZE + ty;
        int B_col = col;

        // Load A tile with pitch
        if (A_row < N && A_col < N) {
            tile_A[ty][tx] = A[A_row * pitch_A + A_col];  // Use pitch
        } else {
            tile_A[ty][tx] = 0.0f;
        }

        // Load B tile (regular)
        if (B_row < N && B_col < N) {
            tile_B[ty][tx] = B[B_row * N + B_col];
        } else {
            tile_B[ty][tx] = 0.0f;
        }

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum = __fmaf_rn(tile_A[ty][k], tile_B[k][tx], sum);
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

void launch_tiled_pitched(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads, int pitch_A) {
    matmul_tiled_pitch<<<blocks, threads>>>(d_A, d_B, d_C, n, pitch_A);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
}

// Add this wrapper function to gemms.cu
void launch_tiled_pitched_wrapper(float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Use global pitch value set in benchmark
    launch_tiled_pitched(d_A, d_B, d_C, n, blocks, threads, g_pitch_A);
}

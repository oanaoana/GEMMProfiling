#include "benchmark.h"
#include "gemms.cuh"
#include "utils.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>

// Define available tests (must match NUM_TESTS in header)
TestCase available_tests[NUM_TESTS] = {
    {"naive", launch_naive, true},
    {"tiled", launch_tiled, true},
    {"cublas", launch_cublas, true}
};

// Define available sizes
const int SIZES[] = {256, 512, 1024, 2048};
const int NUM_SIZES = 4;

// Benchmark function
void runBenchmark(const char* name, int n, KernelFunc kernel,
                  float* h_A, float* h_B, float* h_C,
                  float* d_A, float* d_B, float* d_C,
                  FILE* dataFile) {

    size_t size = n * n * sizeof(float);
    size_t mem_access = 3 * size; // Read A, Read B, Write C
    double operations = 2.0 * n * n * n; // 2*N^3 FLOPs for matrix multiplication
    double arithmetic_intensity = operations / mem_access;

    // Set dimensions
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // For tiled implementation, use TILE_SIZE
    if (strcmp(name, "tiled") == 0) {
        threadsPerBlock = dim3(TILE_SIZE, TILE_SIZE);
        numBlocks = dim3((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);
    }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Warmup run
    kernel(d_A, d_B, d_C, n, numBlocks, threadsPerBlock);
    hipDeviceSynchronize();

    // Clear result matrix
    hipMemset(d_C, 0, size);

    // Timing run
    hipEventRecord(start);
    kernel(d_A, d_B, d_C, n, numBlocks, threadsPerBlock);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    double gigaFlops = (operations / (milliseconds / 1000.0)) / 1e9;
    double bandwidth_gbps = (mem_access / (milliseconds / 1000.0)) / 1e9;

    printf("%s (N=%d): %.2f ms, %.2f GFLOP/s, %.2f GB/s, AI=%.2f\n",
           name, n, milliseconds, gigaFlops, bandwidth_gbps, arithmetic_intensity);

    // Save to CSV
    fprintf(dataFile, "%s,%d,%.2f,%.2f,%.2f,%.2f\n",
            name, n, milliseconds, gigaFlops, bandwidth_gbps, arithmetic_intensity);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Main benchmark function
void runAllBenchmarks(bool* enabled_tests, bool* enabled_sizes) {
    printf("=== Starting Benchmarks ===\n");

    // Open data file for roofline model
    FILE* dataFile = fopen("roofline_data.csv", "w");
    if (!dataFile) {
        printf("ERROR: Could not create roofline_data.csv\n");
        return;
    }
    fprintf(dataFile, "algorithm,size,time_ms,gflops,bandwidth_gb,arithmetic_intensity\n");

    // Test each matrix size
    for (int i = 0; i < NUM_SIZES; i++) {
        if (!enabled_sizes[i]) continue;

        int n = SIZES[i];
        printf("\n--- Testing matrix size %d x %d ---\n", n, n);

        // Allocate host memory
        size_t size = n * n * sizeof(float);
        float *h_A = (float*)malloc(size);
        float *h_B = (float*)malloc(size);
        float *h_C = (float*)malloc(size);

        if (!h_A || !h_B || !h_C) {
            printf("ERROR: Failed to allocate host memory\n");
            return;
        }

        fill_matrix(h_A, n);
        fill_matrix(h_B, n);

        // Allocate device memory
        float *d_A, *d_B, *d_C;
        hipError_t err;
        err = hipMalloc(&d_A, size);
        if (err != hipSuccess) {
            printf("ERROR: hipMalloc d_A failed: %s\n", hipGetErrorString(err));
            return;
        }
        err = hipMalloc(&d_B, size);
        if (err != hipSuccess) {
            printf("ERROR: hipMalloc d_B failed: %s\n", hipGetErrorString(err));
            return;
        }
        err = hipMalloc(&d_C, size);
        if (err != hipSuccess) {
            printf("ERROR: hipMalloc d_C failed: %s\n", hipGetErrorString(err));
            return;
        }

        // Copy input data to device
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        // Run each enabled test
        for (int j = 0; j < NUM_TESTS; j++) {
            if (!enabled_tests[j]) continue;

            printf("\n===== %s =====\n", available_tests[j].name);
            runBenchmark(available_tests[j].name, n, available_tests[j].kernel,
                         h_A, h_B, h_C, d_A, d_B, d_C, dataFile);

            // Verify results for first size only
            if (i == 0) {
                hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
                verify_result(h_A, h_B, h_C, n);
            }
        }

        // Free memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);
        free(h_C);
    }

    fclose(dataFile);
    printf("\nBenchmark complete. Results saved to roofline_data.csv\n");
}
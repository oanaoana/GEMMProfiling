#include "hip/hip_runtime.h"
// generate_test_matrix.cu - Matrix generation and file I/O for numerical analysis
#include "../include/generate_test_matrix.cuh"
#include "../include/utils.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <hiprand.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <time.h>

// Helper function to check if file exists
bool file_exists(const char* filename) {
    struct stat buffer;
    return (stat(filename, &buffer) == 0);
}

// Generate filename for matrix based on type and size
void generate_matrix_filename(char* filename, size_t filename_size, MatrixType type, int n) {
    const char* type_names[] = {
        "wellcond", "illcond", "normaldist", "scaledftz", "skewmag", "fromfile"
    };

    int type_index = (int)type;
    if (type_index < 0 || type_index >= sizeof(type_names)/sizeof(type_names[0])) {
        type_index = 0; // fallback to random
    }

    snprintf(filename, filename_size, "data/matrix_%s_%dx%d.bin", type_names[type_index], n, n);
}

// Write matrix to binary file
bool write_matrix_to_file(const char* filename, float* matrix, int n) {
    FILE* fp = fopen(filename, "wb");
    if (!fp) {
        printf("ERROR: Cannot create matrix file %s\n", filename);
        return false;
    }

    // Write header with matrix size for verification
    int header[2] = {n, n};
    if (fwrite(header, sizeof(int), 2, fp) != 2) {
        printf("ERROR: Failed to write matrix header to %s\n", filename);
        fclose(fp);
        return false;
    }

    // Write matrix data
    size_t elements_written = fwrite(matrix, sizeof(float), n * n, fp);
    fclose(fp);

    if (elements_written != n * n) {
        printf("ERROR: Failed to write complete matrix to %s\n", filename);
        return false;
    }

    printf("Matrix saved to: %s\n", filename);
    return true;
}

// Read matrix from binary file
bool read_matrix_from_file(const char* filename, float* matrix, int n) {
    FILE* fp = fopen(filename, "rb");
    if (!fp) {
        printf("ERROR: Cannot open matrix file %s\n", filename);
        return false;
    }

    // Read and verify header
    int header[2];
    if (fread(header, sizeof(int), 2, fp) != 2) {
        printf("ERROR: Failed to read matrix header from %s\n", filename);
        fclose(fp);
        return false;
    }

    if (header[0] != n || header[1] != n) {
        printf("ERROR: Matrix size mismatch in %s. Expected %dx%d, got %dx%d\n",
               filename, n, n, header[0], header[1]);
        fclose(fp);
        return false;
    }

    // Read matrix data
    size_t elements_read = fread(matrix, sizeof(float), n * n, fp);
    fclose(fp);

    if (elements_read != n * n) {
        printf("ERROR: Failed to read complete matrix from %s\n", filename);
        return false;
    }

    printf("Matrix loaded from: %s\n", filename);
    return true;
}

// SVD-based matrix generation with controlled condition number and custom seed
void generate_matrix_svd_with_seed(float* d_A, int n, float cond_num, unsigned long long seed) {
    // Create cuBLAS and cuSOLVER handles
    hipblasHandle_t cublasH;
    hipsolverHandle_t cusolverH;
    hipblasCreate(&cublasH);
    hipsolverDnCreate(&cusolverH);

    // Allocate device memory
    float *d_Q1, *d_Q2, *d_Rwork;
    float *d_temp1, *d_temp2;
    hipMalloc(&d_Q1, n * n * sizeof(float));
    hipMalloc(&d_Q2, n * n * sizeof(float));
    hipMalloc(&d_temp1, n * n * sizeof(float));
    hipMalloc(&d_temp2, n * n * sizeof(float));

    // Step 1: Generate two random matrices
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerateUniform(gen, d_temp1, n * n);
    hiprandGenerateUniform(gen, d_temp2, n * n);

    // Step 2: QR decomposition → Q1, Q2
    std::vector<int> devInfo(1);
    int *d_info;
    hipMalloc(&d_info, sizeof(int));
    int lwork = 0;
    hipsolverDnSgeqrf_bufferSize(cusolverH, n, n, d_temp1, n, &lwork);
    hipMalloc(&d_Rwork, lwork * sizeof(float));
    std::vector<float> tau(n);
    float* d_tau;
    hipMalloc(&d_tau, n * sizeof(float));

    // QR on d_temp1 → d_Q1
    hipsolverDnSgeqrf(cusolverH, n, n, d_temp1, n, d_tau, d_Rwork, lwork, d_info);
    hipsolverDnSorgqr(cusolverH, n, n, n, d_temp1, n, d_tau, d_Rwork, lwork, d_info);
    hipMemcpy(d_Q1, d_temp1, n * n * sizeof(float), hipMemcpyDeviceToDevice);

    // QR on d_temp2 → d_Q2
    hipsolverDnSgeqrf(cusolverH, n, n, d_temp2, n, d_tau, d_Rwork, lwork, d_info);
    hipsolverDnSorgqr(cusolverH, n, n, n, d_temp2, n, d_tau, d_Rwork, lwork, d_info);
    hipMemcpy(d_Q2, d_temp2, n * n * sizeof(float), hipMemcpyDeviceToDevice);

    // Step 3: Construct Sigma on host and upload
    std::vector<float> h_sigma(n * n, 0.0f);
    for (int i = 0; i < n; ++i) {
        float sval = std::pow(cond_num, -((float)i / (n - 1)));  // log-uniform decay
        h_sigma[i * n + i] = sval;
    }
    float* d_sigma;
    hipMalloc(&d_sigma, n * n * sizeof(float));
    hipMemcpy(d_sigma, h_sigma.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

    // Step 4: A_illcond = Q1 * Sigma * Q2^T
    float alpha = 1.0f, beta = 0.0f;
    float* d_tmp;
    hipMalloc(&d_tmp, n * n * sizeof(float));
    // tmp = Q1 * Sigma
    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_Q1, n, d_sigma, n, &beta, d_tmp, n);
    // d_A = tmp * Q2^T
    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, n, &alpha, d_tmp, n, d_Q2, n, &beta, d_A, n);

    // Cleanup
    hipFree(d_Q1); hipFree(d_Q2); hipFree(d_temp1); hipFree(d_temp2);
    hipFree(d_sigma); hipFree(d_tmp); hipFree(d_tau); hipFree(d_Rwork); hipFree(d_info);
    hipblasDestroy(cublasH); hipsolverDnDestroy(cusolverH); hiprandDestroyGenerator(gen);
}

// CUDA kernel for scaling matrix elements
__global__ void scale_matrix_kernel(float* data, int size, float scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] *= scale;
    }
}

// CUDA kernel to convert uniform [0,1) values to Rademacher {-1, +1}
__global__ void rademacher_transform_kernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Convert uniform [0,1) to Rademacher: -1 if < 0.5, +1 if >= 0.5
        data[idx] = (data[idx] < 0.5f) ? -1.0f : 1.0f;
    }
}

// CUDA kernel to convert integer {0,1} signs directly to Rademacher {-1, +1}
// Unified CUDA kernel to apply random signs from integer {0,1} to {-1,+1}
// If replace_mode=true: result[i] = sign(signs[i])
// If replace_mode=false: result[i] *= sign(signs[i])
__global__ void random_signs_kernel(int* signs, float* result, int size, bool replace_mode) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Convert integer {0,1} to sign {-1,+1}
        float sign = (signs[idx] == 0) ? -1.0f : 1.0f;

        if (replace_mode) {
            // Replace mode: result = sign
            result[idx] = sign;
        } else {
            // Multiply mode: result *= sign
            result[idx] *= sign;
        }
    }
}

// CUDA kernel to generate jittered Rademacher: sign * (1 + δ) where δ ∈ (-2^(-12), 2^(-12))
__global__ void rademacher_jittered_kernel(int* signs, float* jitter, float* result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Convert integer {0,1} to sign {-1,+1}
        float sign = (signs[idx] == 0) ? -1.0f : 1.0f;

        // Apply jitter: sign * (1 + δ) where δ ∈ (-2^(-12), 2^(-12))
        result[idx] = sign * (1.0f + jitter[idx]);
    }
}

// CUDA kernel to generate 2-powers matrix: s * 2^(-p) where s is ±1 and p is integer [10,30]
__global__ void twopowers_transform_kernel(int* signs, int* exponents, float* result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Use sign directly: convert 0 to -1, 1 to +1
        float s = (signs[idx] == 0) ? -1.0f : 1.0f;

        // Use the integer exponent directly (should be in range [10,30])
        int p = exponents[idx];

        // Compute s * 2^(-p) using ldexpf
        result[idx] = s * ldexpf(1.0f, -p);
    }
}// CUDA kernel for clamping matrix elements to interval (min_val, max_val)
// CUDA kernel to transform values from interval [a,b] to [c,d]
// Formula: y = c + (x - a) * (d - c) / (b - a)
__global__ void transform_interval_kernel(float* data, int n, float a, float b, float c, float d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = data[idx];
        // Transform from [a,b] to [c,d]
        float scale = (d - c) / (b - a);
        data[idx] = c + (x - a) * scale;
    }
}

// Simple kernel for integer range transformation
__global__ void transform_int_range_kernel(int* data, int size, int min_val, int range) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        unsigned int val = ((unsigned int*)data)[idx];
        data[idx] = min_val + (val % range);
    }
}

// Generate integer uniform distribution in range [min_val, max_val] (inclusive)
void generate_integer_uniform_with_seed(int* d_integers, int m, int n, int min_val, int max_val, unsigned long long seed) {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);

    size_t total_elements = m * n;
    hiprandGenerate(gen, (unsigned int*)d_integers, total_elements);

    // Transform to desired range [min_val, max_val]
    int range = max_val - min_val + 1;
    dim3 threads(256);
    dim3 blocks((total_elements + threads.x - 1) / threads.x);

    // Launch kernel to map to range
    transform_int_range_kernel<<<blocks, threads>>>(d_integers, total_elements, min_val, range);
    hipDeviceSynchronize();

    hiprandDestroyGenerator(gen);
}

// Non-seeded version using time(NULL)
void generate_integer_uniform(int* d_integers, int m, int n, int min_val, int max_val) {
    generate_integer_uniform_with_seed(d_integers, m, n, min_val, max_val, time(NULL));
}

// Efficient matrix generation for multi-sample analysis
// - Works directly on pre-allocated device memory
// - No file I/O overhead
// - No memory allocation/deallocation
// - Supports all matrix types with custom seeds
void generate_matrix_device_with_seed(float* d_matrix, int n, MatrixType type, unsigned long long seed) {
    switch (type) {
        case MATRIX_ODO_WELL_CONDITIONED:
            generate_matrix_svd_with_seed(d_matrix, n, WELL_COND_NUMBER, seed);
            break;

        case MATRIX_ODO_ILL_CONDITIONED:
            generate_matrix_svd_with_seed(d_matrix, n, ILL_COND_NUMBER, seed);
            break;

        case MATRIX_ZEROMEAN:
            {
                // Generate normal distribution with zero mean and std = 1/sqrt(n)
                // This gives entries with expected magnitude scaling appropriately with matrix size
                const float mean = 0.0f;
                const float std = 1.0f / sqrtf((float)n);
                generate_matrix_distribution_with_seed(d_matrix, n, n, DIST_NORMAL, mean, std, seed);
            }
            break;

        case MATRIX_UNIFORM_POSITIVE:
            {
                // Generate uniform distribution in (0,1) interval
                const float epsilon = 1e-6f;  // Small margin for open interval
                generate_matrix_distribution_with_seed(d_matrix, n, n, DIST_UNIFORM,
                                                      epsilon, 1.0f - epsilon, seed);
            }
            break;

        case MATRIX_RADEMACHER:
            {
                // Generate jittered Rademacher distribution: sign * (1 + δ) where δ ∈ (-2^(-12), 2^(-12))
                // This adds small perturbations to the ±1 structure for more realistic numerical analysis
                int total_elements = n * n;

                // Allocate temporary memory for signs and jitter
                int *d_signs;
                float *d_jitter;
                hipMalloc(&d_signs, total_elements * sizeof(int));
                hipMalloc(&d_jitter, total_elements * sizeof(float));

                // Generate integer signs {0,1}
                generate_integer_uniform_with_seed(d_signs, n, n, 0, 1, seed);

                // Generate jitter δ ∈ (-2^(-12), 2^(-12)) = (-1/4096, 1/4096)
                const float jitter_bound = 1.0f / 4096.0f;  // 2^(-12)
                generate_matrix_distribution_with_seed(d_jitter, n, n, DIST_UNIFORM,
                                                      -jitter_bound, jitter_bound, seed + 54321);

                // Convert to jittered Rademacher: sign * (1 + δ)
                int block_size = 256;
                int grid_size = (total_elements + block_size - 1) / block_size;

                rademacher_jittered_kernel<<<grid_size, block_size>>>(d_signs, d_jitter, d_matrix, total_elements);
                hipDeviceSynchronize();

                // Cleanup temporary memory
                hipFree(d_signs);
                hipFree(d_jitter);
            }
            break;

        case MATRIX_SANITY:
            {
                // Generate original Rademacher distribution: exact ±1 values
                // Perfect for debugging - should always produce zero errors with exact arithmetic
                int total_elements = n * n;

                // Allocate temporary memory for signs
                int *d_signs;
                hipMalloc(&d_signs, total_elements * sizeof(int));

                // Generate integer signs {0,1}
                generate_integer_uniform_with_seed(d_signs, n, n, 0, 1, seed);

                // Convert directly to exact Rademacher {-1, +1}
                int block_size = 256;
                int grid_size = (total_elements + block_size - 1) / block_size;

                random_signs_kernel<<<grid_size, block_size>>>(d_signs, d_matrix, total_elements, true);
                hipDeviceSynchronize();

                // Cleanup temporary memory
                hipFree(d_signs);
            }
            break;

        case MATRIX_SCALED_2POWERS:
            {
                // Generate 2-powers matrix: s * 2^(-p) where s is ±1 and p is integer [10,30]
                int total_elements = n * n;

                // Allocate temporary memory for signs and exponents
                int *d_signs, *d_exponents;
                hipMalloc(&d_signs, total_elements * sizeof(int));
                hipMalloc(&d_exponents, total_elements * sizeof(int));

                // Generate integer distributions for signs {0,1} and exponents [10,30]
                generate_integer_uniform_with_seed(d_signs, n, n, 0, 1, seed);
                generate_integer_uniform_with_seed(d_exponents, n, n, 10, 30, seed + 12345);

                // Transform to 2-powers matrix
                int block_size = 256;
                int grid_size = (total_elements + block_size - 1) / block_size;

                twopowers_transform_kernel<<<grid_size, block_size>>>(d_signs, d_exponents, d_matrix, total_elements);
                hipDeviceSynchronize();

                // Cleanup temporary memory
                hipFree(d_signs);
                hipFree(d_exponents);
            }
            break;

        case MATRIX_LOGNORMAL:
            {
                // Generate signed log-normal distribution: sign * exp(N(0, σ)) where σ ∈ {1, 2}
                // Random signs ensure zero mean while keeping log-normal magnitude distribution
                int total_elements = n * n;
                const float sigma = (n < 1024) ? 1.0f : 2.0f;
                const float mean = 0.0f;  // Normal distribution mean = 0

                // Allocate temporary memory for signs
                int *d_signs;
                hipMalloc(&d_signs, total_elements * sizeof(int));

                // Generate positive log-normal values: exp(N(0, σ))
                generate_matrix_distribution_with_seed(d_matrix, n, n, DIST_LOG_NORMAL, mean, sigma, seed);

                // Generate random signs {0,1} → {-1,+1}
                generate_integer_uniform_with_seed(d_signs, n, n, 0, 1, seed + 98765);

                // Apply random signs to get zero-mean signed log-normal
                int block_size = 256;
                int grid_size = (total_elements + block_size - 1) / block_size;

                random_signs_kernel<<<grid_size, block_size>>>(d_signs, d_matrix, total_elements, false);
                hipDeviceSynchronize();

                // Cleanup temporary memory
                hipFree(d_signs);
            }
            break;

        case MATRIX_FROM_FILE:
            printf("ERROR: MATRIX_FROM_FILE not supported in generate_matrix_device_with_seed\n");
            printf("Use get_matrix() for file-based matrix loading\n");
            // Fill with zeros as fallback
            hipMemset(d_matrix, 0, n * n * sizeof(float));
            break;

        default:
            printf("Unknown matrix type %d, using moderate SVD conditioning\n", (int)type);
            generate_matrix_svd_with_seed(d_matrix, n, 10.0f, seed);
            break;
    }
}

// SVD-based matrix generation with controlled condition number
void generate_matrix_svd(float* d_A, int n, float cond_num) {
    // Use current time as seed for better randomization
    unsigned long long seed = (unsigned long long)time(NULL) + (unsigned long long)clock();
    generate_matrix_svd_with_seed(d_A, n, cond_num, seed);
}

// Main function to get matrix (from cache or generate new)
bool get_matrix(float* matrix, int n, MatrixType type, const char* custom_filename) {
    char filename[512];

    // Use custom filename if provided and type is MATRIX_FROM_FILE
    if (type == MATRIX_FROM_FILE && custom_filename) {
        strncpy(filename, custom_filename, sizeof(filename) - 1);
        filename[sizeof(filename) - 1] = '\0';
    } else {
        // Generate standard filename
        generate_matrix_filename(filename, sizeof(filename), type, n);
    }

    // Check if file exists and try to load it
    if (file_exists(filename)) {
        printf("Found existing matrix file: %s\n", filename);
        if (read_matrix_from_file(filename, matrix, n)) {
            return true;
        } else {
            printf("Failed to load matrix from file, regenerating...\n");
        }
    }

    // File doesn't exist or failed to load, generate new matrix
    printf("Generating new matrix (type: %d, size: %dx%d)\n", (int)type, n, n);

    if (type == MATRIX_FROM_FILE && custom_filename) {
        printf("ERROR: Custom file %s not found and cannot generate matrix for MATRIX_FROM_FILE type\n", custom_filename);
        return false;
    }

    // Allocate device memory for matrix generation
    float* d_matrix;
    hipMalloc(&d_matrix, n * n * sizeof(float));

    // Generate matrix based on type
    switch (type) {
        case MATRIX_ODO_WELL_CONDITIONED:
            printf("Generating well-conditioned matrix using SVD (condition number: %.2e)\n", WELL_COND_NUMBER);
            generate_matrix_svd(d_matrix, n, WELL_COND_NUMBER);
            break;

        case MATRIX_ODO_ILL_CONDITIONED:
            printf("Generating ill-conditioned matrix using SVD (condition number: %.2e)\n", ILL_COND_NUMBER);
            generate_matrix_svd(d_matrix, n, ILL_COND_NUMBER);
            break;

        case MATRIX_ZEROMEAN:
            printf("Generating zero-mean normal distribution matrix N(0, 1/sqrt(n)) with std=%.6f\n", 1.0f/sqrtf((float)n));
            generate_matrix_distribution(d_matrix, n, n, DIST_NORMAL, 0.0f, 1.0f/sqrtf((float)n));
            break;

        case MATRIX_UNIFORM_POSITIVE:
            printf("Generating uniform distribution matrix in (0,1) interval\n");
            {
                const float epsilon = 1e-6f;
                generate_matrix_distribution(d_matrix, n, n, DIST_UNIFORM, epsilon, 1.0f - epsilon);
            }
            break;

        case MATRIX_RADEMACHER:
            printf("Generating jittered Rademacher distribution matrix: sign * (1 + δ), δ ∈ (-2^(-12), 2^(-12))\n");
            {
                // Generate jittered Rademacher for more realistic numerical behavior
                int total_elements = n * n;

                // Allocate temporary memory for signs and jitter
                int *d_signs;
                float *d_jitter;
                hipMalloc(&d_signs, total_elements * sizeof(int));
                hipMalloc(&d_jitter, total_elements * sizeof(float));

                // Generate integer signs {0,1}
                generate_integer_uniform(d_signs, n, n, 0, 1);

                // Generate jitter δ ∈ (-2^(-12), 2^(-12)) = (-1/4096, 1/4096)
                const float jitter_bound = 1.0f / 4096.0f;  // 2^(-12)
                generate_matrix_distribution(d_jitter, n, n, DIST_UNIFORM,
                                           -jitter_bound, jitter_bound);

                // Convert to jittered Rademacher: sign * (1 + δ)
                int block_size = 256;
                int grid_size = (total_elements + block_size - 1) / block_size;

                rademacher_jittered_kernel<<<grid_size, block_size>>>(d_signs, d_jitter, d_matrix, total_elements);
                hipDeviceSynchronize();

                // Cleanup temporary memory
                hipFree(d_signs);
                hipFree(d_jitter);
            }
            break;

        case MATRIX_SANITY:
            printf("Generating SANITY matrix: exact Rademacher ±1 (for debugging/verification)\n");
            {
                // Generate original exact Rademacher - perfect for sanity checks
                int total_elements = n * n;

                // Allocate temporary memory for signs
                int *d_signs;
                hipMalloc(&d_signs, total_elements * sizeof(int));

                // Generate integer signs {0,1}
                generate_integer_uniform(d_signs, n, n, 0, 1);

                // Convert directly to exact Rademacher {-1, +1}
                int block_size = 256;
                int grid_size = (total_elements + block_size - 1) / block_size;

                random_signs_kernel<<<grid_size, block_size>>>(d_signs, d_matrix, total_elements, true);
                hipDeviceSynchronize();

                // Cleanup temporary memory
                hipFree(d_signs);
            }
            break;

        case MATRIX_SCALED_2POWERS:
            {
                printf("Generating 2-powers matrix: s * 2^(-p) where s=±1, p∈[10,30]\n");
                int total_elements = n * n;

                // Allocate temporary memory for signs and exponents
                int *d_signs, *d_exponents;
                hipMalloc(&d_signs, total_elements * sizeof(int));
                hipMalloc(&d_exponents, total_elements * sizeof(int));

                // Generate integer distributions for signs {0,1} and exponents [10,30]
                generate_integer_uniform(d_signs, n, n, 0, 1);
                generate_integer_uniform(d_exponents, n, n, 10, 30);

                // Transform to 2-powers matrix
                int block_size = 256;
                int grid_size = (total_elements + block_size - 1) / block_size;

                twopowers_transform_kernel<<<grid_size, block_size>>>(d_signs, d_exponents, d_matrix, total_elements);
                hipDeviceSynchronize();

                // Cleanup temporary memory
                hipFree(d_signs);
                hipFree(d_exponents);
                break;
            }

        case MATRIX_LOGNORMAL:
            {
                // Generate signed log-normal distribution: sign * exp(N(0, σ)) where σ ∈ {1, 2}
                // Random signs ensure zero mean while keeping log-normal magnitude distribution
                int total_elements = n * n;
                const float sigma = (n < 1024) ? 1.0f : 2.0f;
                const float mean = 0.0f;  // Normal distribution mean = 0

                printf("Generating signed log-normal distribution matrix: sign * exp(N(0, %.1f))\n", sigma);

                // Allocate temporary memory for signs
                int *d_signs;
                hipMalloc(&d_signs, total_elements * sizeof(int));

                // Generate positive log-normal values: exp(N(0, σ))
                generate_matrix_distribution(d_matrix, n, n, DIST_LOG_NORMAL, mean, sigma);

                // Generate random signs {0,1} → {-1,+1} (use deterministic seed based on matrix address)
                uintptr_t address_seed = reinterpret_cast<uintptr_t>(d_matrix);
                generate_integer_uniform_with_seed(d_signs, n, n, 0, 1, static_cast<unsigned long long>(address_seed));

                // Apply random signs to get zero-mean signed log-normal
                int block_size = 256;
                int grid_size = (total_elements + block_size - 1) / block_size;

                random_signs_kernel<<<grid_size, block_size>>>(d_signs, d_matrix, total_elements, false);
                hipDeviceSynchronize();

                // Cleanup temporary memory
                hipFree(d_signs);
            }
            break;

        case MATRIX_FROM_FILE:
            printf("ERROR: MATRIX_FROM_FILE should not reach generation code\n");
            hipFree(d_matrix);
            return false;

        default:
            printf("Unknown matrix type %d, using moderate SVD conditioning\n", (int)type);
            generate_matrix_svd(d_matrix, n, 10.0f);
            break;
    }

    // Copy result back to host
    hipMemcpy(matrix, d_matrix, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup device memory
    hipFree(d_matrix);

    // Save generated matrix to file for future use
    if (!write_matrix_to_file(filename, matrix, n)) {
        printf("Warning: Failed to save matrix to file, but matrix was generated successfully\n");
    }

    return true;
}

// Utility function to print matrix statistics
void print_matrix_stats(float* matrix, int n, const char* name) {
    double sum = 0.0, sum_sq = 0.0;
    float min_val = matrix[0], max_val = matrix[0];

    for (int i = 0; i < n * n; i++) {
        float val = matrix[i];
        sum += val;
        sum_sq += val * val;
        if (val < min_val) min_val = val;
        if (val > max_val) max_val = val;
    }

    double mean = sum / (n * n);
    double variance = (sum_sq / (n * n)) - (mean * mean);
    double frobenius_norm = sqrt(sum_sq);

    printf("Matrix %s statistics:\n", name);
    printf("  Size: %dx%d\n", n, n);
    printf("  Range: [%.6e, %.6e]\n", min_val, max_val);
    printf("  Mean: %.6e\n", mean);
    printf("  Std dev: %.6e\n", sqrt(variance));
    printf("  Frobenius norm: %.6e\n", frobenius_norm);
}

// Generate matrix with specified distribution (operates on GPU memory)
// Parameters:
//   d_matrix: Device memory pointer (must be pre-allocated)
//   m, n: Matrix dimensions (rows, columns)
//   dist_type: DIST_UNIFORM, DIST_NORMAL, or DIST_LOG_NORMAL
//   param1, param2: Distribution parameters
//     - UNIFORM: param1=min, param2=max
//     - NORMAL: param1=mean, param2=std_dev
//     - LOG_NORMAL: param1=log_mean, param2=log_std_dev

// Seeded version for multi-sample analysis
void generate_matrix_distribution_with_seed(float* d_matrix, int m, int n, DistributionType dist_type,
                                           float param1, float param2, unsigned long long seed) {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);

    size_t total_elements = m * n;

    switch (dist_type) {
        case DIST_UNIFORM:
            hiprandGenerateUniform(gen, d_matrix, total_elements);
            // Transform from [0,1) to [param1, param2)
            if (param1 != 0.0f || param2 != 1.0f) {
                dim3 threads(256);
                dim3 blocks((total_elements + threads.x - 1) / threads.x);
                transform_interval_kernel<<<blocks, threads>>>(
                    d_matrix, total_elements,
                    0.0f, 1.0f,      // from [0, 1)
                    param1, param2   // to [param1, param2)
                );
                hipDeviceSynchronize();
            }
            break;

        case DIST_NORMAL:
            hiprandGenerateNormal(gen, d_matrix, total_elements, param1, param2);
            break;

        case DIST_LOG_NORMAL:
            hiprandGenerateLogNormal(gen, d_matrix, total_elements, param1, param2);
            break;

        default:
            printf("Unknown distribution type %d, using uniform [0,1)\n", (int)dist_type);
            hiprandGenerateUniform(gen, d_matrix, total_elements);
            break;
    }

    hiprandDestroyGenerator(gen);
}

// Original version (uses time(NULL) for seed)
void generate_matrix_distribution(float* d_matrix, int m, int n, DistributionType dist_type,
                                 float param1, float param2) {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    size_t total_elements = m * n;

    switch (dist_type) {
        case DIST_UNIFORM:
            hiprandGenerateUniform(gen, d_matrix, total_elements);
            // Transform from [0,1) to [param1, param2)
            if (param1 != 0.0f || param2 != 1.0f) {
                dim3 threads(256);
                dim3 blocks((total_elements + threads.x - 1) / threads.x);
                transform_interval_kernel<<<blocks, threads>>>(
                    d_matrix, total_elements,
                    0.0f, 1.0f,      // from [0, 1)
                    param1, param2   // to [param1, param2)
                );
                hipDeviceSynchronize();
            }
            break;

        case DIST_NORMAL:
            hiprandGenerateNormal(gen, d_matrix, total_elements, param1, param2);
            break;

        case DIST_LOG_NORMAL:
            hiprandGenerateLogNormal(gen, d_matrix, total_elements, param1, param2);
            break;

        default:
            printf("Unknown distribution type %d, using uniform [0,1)\n", (int)dist_type);
            hiprandGenerateUniform(gen, d_matrix, total_elements);
            break;
    }

    hiprandDestroyGenerator(gen);
}

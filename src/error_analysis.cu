#include "hip/hip_runtime.h"
// error_analysis.cu - Consolidated error analysis functionality
#include "../include/error_analysis.cuh"
#include "../include/config.h"  // For configuration constants and SIZES
#include "../include/generate_test_matrix.cuh"  // For get_matrix and print_matrix_stats
#include "../include/gemms.cuh"
#include "../include/utils.cuh"
#include "../include/matrix_utils.cuh"  // For matrix utility functions
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

// Kernel to analyze round-off errors during tiled GEMM
__global__ void analyze_tiled_gemm_errors(
    float *A, float *B, float *C_result, float *C_reference,
    int N,
    float *tile_norms, float *tile_condition_numbers,
    float *accumulated_errors, int *error_counts) {

    // Shared memory for tiles
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE + 1];
    //__shared__ float partial_results[TILE_SIZE][TILE_SIZE + 1];

    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float sum = 0.0f;
    float error_accumulation = 0.0f;
    int num_tiles = (N + TILE_SIZE - 1) / TILE_SIZE;

    // Global tile index for this block
    int tile_idx = by * gridDim.x + bx;

    for (int t = 0; t < num_tiles; ++t) {
        // Load tiles (same as original tiled implementation)
        int A_row = row;
        int A_col = t * TILE_SIZE + tx;
        tile_A[ty][tx] = (A_row < N && A_col < N) ? A[A_row * N + A_col] : 0.0f;

        int B_row = t * TILE_SIZE + ty;
        int B_col = col;
        tile_B[ty][tx] = (B_row < N && B_col < N) ? B[B_row * N + B_col] : 0.0f;

        __syncthreads();

        // Compute Frobenius norm of current tiles (thread 0 only)
        if (tx == 0 && ty == 0) {
            float norm_A = compute_frobenius_norm_tile((float*)tile_A, TILE_SIZE, TILE_SIZE);
            float norm_B = compute_frobenius_norm_tile((float*)tile_B, TILE_SIZE, TILE_SIZE);

            // Compute condition numbers for the tiles
            float cond_A = estimate_condition_number_tile((float*)tile_A, TILE_SIZE, TILE_SIZE);
            float cond_B = estimate_condition_number_tile((float*)tile_B, TILE_SIZE, TILE_SIZE);

            // Store norms (using atomic add for accumulation across tiles)
            atomicAdd(&tile_norms[tile_idx * 2], norm_A);
            atomicAdd(&tile_norms[tile_idx * 2 + 1], norm_B);

            // Store condition numbers (using atomic add for averaging later)
            atomicAdd(&tile_condition_numbers[tile_idx * 2], cond_A);
            atomicAdd(&tile_condition_numbers[tile_idx * 2 + 1], cond_B);
        }

        // Compute partial dot product with error tracking
        float old_sum = sum;
        for (int k = 0; k < TILE_SIZE; ++k) {
            float product = tile_A[ty][k] * tile_B[k][tx];
            sum += product;

            // Track numerical error accumulation
            float new_error = fabsf(sum - old_sum - product);
            error_accumulation += new_error;
            old_sum = sum;
        }

        __syncthreads();
    }

    // Store final result and error analysis
    if (row < N && col < N) {
        C_result[row * N + col] = sum;

        // Compute error compared to reference
        float reference_val = C_reference[row * N + col];
        float absolute_error = fabsf(sum - reference_val);
        float relative_error = (fabsf(reference_val) > 1e-10f) ?
                              absolute_error / fabsf(reference_val) : absolute_error;

        // Store error data
        int idx = row * N + col;
        accumulated_errors[idx * 3] = absolute_error;
        accumulated_errors[idx * 3 + 1] = relative_error;
        accumulated_errors[idx * 3 + 2] = error_accumulation;

        // Count significant errors
        if (relative_error > 1e-6f) {
            atomicAdd(error_counts, 1);
        }
    }
}

// Host function to run comprehensive numerical analysis
void run_numerical_analysis(float* h_A, float* h_B, int n, const char* output_filename) {
    printf("\n=== Numerical Analysis of Tiled GEMM ===\n");
    printf("Matrix size: %d x %d\n", n, n);
    printf("Using original matrices without standardization.\n");

    size_t size = n * n * sizeof(float);
    size_t tile_data_size = ((n + TILE_SIZE - 1) / TILE_SIZE) *
                           ((n + TILE_SIZE - 1) / TILE_SIZE) * 2 * sizeof(float);
    size_t condition_data_size = ((n + TILE_SIZE - 1) / TILE_SIZE) *
                                ((n + TILE_SIZE - 1) / TILE_SIZE) * 2 * sizeof(float);

    // Allocate device memory
    float *d_A, *d_B, *d_C_tiled, *d_C_reference;
    float *d_tile_norms, *d_condition_numbers, *d_accumulated_errors;
    int *d_error_counts;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C_tiled, size);
    hipMalloc(&d_C_reference, size);
    hipMalloc(&d_tile_norms, tile_data_size);
    hipMalloc(&d_condition_numbers, condition_data_size);
    hipMalloc(&d_accumulated_errors, size * 3); // abs, rel, accumulated
    hipMalloc(&d_error_counts, sizeof(int));

    // Copy input data
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Initialize arrays
    hipMemset(d_tile_norms, 0, tile_data_size);
    hipMemset(d_condition_numbers, 0, condition_data_size);
    hipMemset(d_accumulated_errors, 0, size * 3);
    hipMemset(d_error_counts, 0, sizeof(int));

    // Compute reference result using cuBLAS (high precision)
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alpha = 1.0f, beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
                &alpha, d_B, n, d_A, n, &beta, d_C_reference, n);
    hipblasDestroy(handle);

    // Run analysis kernel
    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);

    analyze_tiled_gemm_errors<<<blocks, threads>>>(
        d_A, d_B, d_C_tiled, d_C_reference, n,
        d_tile_norms, d_condition_numbers, d_accumulated_errors, d_error_counts);

    hipDeviceSynchronize();

    // Retrieve results
    float *h_tile_norms = (float*)malloc(tile_data_size);
    float *h_condition_numbers = (float*)malloc(condition_data_size);
    float *h_accumulated_errors = (float*)malloc(size * 3);
    int h_error_count;

    hipMemcpy(h_tile_norms, d_tile_norms, tile_data_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_condition_numbers, d_condition_numbers, condition_data_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_accumulated_errors, d_accumulated_errors, size * 3, hipMemcpyDeviceToHost);
    hipMemcpy(&h_error_count, d_error_counts, sizeof(int), hipMemcpyDeviceToHost);

    // Analyze and report results
    analyze_numerical_results(h_tile_norms, h_condition_numbers, h_accumulated_errors, n, h_error_count, output_filename);

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C_tiled); hipFree(d_C_reference);
    hipFree(d_tile_norms); hipFree(d_condition_numbers); hipFree(d_accumulated_errors);
    hipFree(d_error_counts);
    free(h_tile_norms); free(h_condition_numbers); free(h_accumulated_errors);
}

// Host function to analyze and report numerical results
void analyze_numerical_results(float* tile_norms, float* condition_numbers, float* errors, int n, int error_count, const char* filename) {
    FILE* fp = fopen(filename, "w");
    if (!fp) {
        printf("ERROR: Cannot create output file %s\n", filename);
        return;
    }

    fprintf(fp, "# Numerical Analysis Results for %dx%d Matrix\n", n, n);
    fprintf(fp, "# Columns: i, j, absolute_error, relative_error, accumulated_error\n");

    double total_abs_error = 0.0;
    double max_rel_error = 0.0;
    double total_accumulated_error = 0.0;

    // Calculate average condition numbers for tiles
    int num_tiles = ((n + TILE_SIZE - 1) / TILE_SIZE) * ((n + TILE_SIZE - 1) / TILE_SIZE);
    double avg_condition_A = 0.0, avg_condition_B = 0.0;

    for (int t = 0; t < num_tiles; t++) {
        avg_condition_A += condition_numbers[t * 2];
        avg_condition_B += condition_numbers[t * 2 + 1];
    }
    avg_condition_A /= num_tiles;
    avg_condition_B /= num_tiles;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            int idx = i * n + j;
            float abs_err = errors[idx * 3];
            float rel_err = errors[idx * 3 + 1];
            float acc_err = errors[idx * 3 + 2];

            fprintf(fp, "%d %d %.10e %.10e %.10e\n", i, j, abs_err, rel_err, acc_err);

            total_abs_error += abs_err;
            max_rel_error = fmaxf(max_rel_error, rel_err);
            total_accumulated_error += acc_err;
        }
    }

    fclose(fp);

    // Print summary statistics including condition numbers
    printf("\n--- Error Analysis Summary ---\n");
    printf("Total elements with significant errors: %d / %d\n", error_count, n * n);
    printf("Average absolute error: %.10e\n", total_abs_error / (n * n));
    printf("Maximum relative error: %.10e\n", max_rel_error);
    printf("Average accumulated error: %.10e\n", total_accumulated_error / (n * n));
    printf("Average condition number (tiles A): %.2e\n", avg_condition_A);
    printf("Average condition number (tiles B): %.2e\n", avg_condition_B);
    printf("Results saved to: %s\n", filename);
}

// Helper function to test different tile sizes
void compare_tile_sizes(float* h_A, float* h_B, int n) {
    printf("\n=== Comparing Different Tile Sizes ===\n");

    // Test different tile sizes by recompiling with different TILE_SIZE values
    // This is a placeholder - in practice you'd want configurable tile sizes

    const int test_tile_sizes[] = {8, 16, 32};
    const int num_tile_sizes = sizeof(test_tile_sizes) / sizeof(test_tile_sizes[0]);

    for (int i = 0; i < num_tile_sizes; i++) {
        printf("Analyzing tile size %d...\n", test_tile_sizes[i]);
        // Note: This would require runtime tile size configuration
        // For now, just report current TILE_SIZE analysis
        char filename[256];
        snprintf(filename, sizeof(filename), "data/numerical_analysis_tile%d_n%d.dat", TILE_SIZE, n);
        run_numerical_analysis(h_A, h_B, n, filename);
    }
}


// Setup matrix data based on type - now uses cached matrix generation
void setupMatrix(float* matrix, int n, MatrixType type, const char* filename) {
    if (!get_matrix(matrix, n, type, filename)) {
        printf("ERROR: Failed to setup matrix of type %d\n", (int)type);
        printf("Falling back to random matrix generation\n");
        fill_matrix(matrix, n);
    }
}

// Run matrix tests with specified configuration
void runMatrixTests(int n, MatrixTestConfig* configs, int num_configs) {
    printf("\n--- Testing matrix size %d x %d ---\n", n, n);

    size_t size = n * n * sizeof(float);

    // Allocate host memory once for all tests
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);

    if (!h_A || !h_B) {
        printf("ERROR: Failed to allocate host memory\n");
        return;
    }

    // Run each enabled test configuration
    for (int i = 0; i < num_configs; i++) {
        if (!configs[i].enabled) continue;

        printf("\n=== Running test: %s ===\n", configs[i].name);
        printf("Description: %s\n", configs[i].description);

        // Setup matrices according to configuration
        printf("Setting up matrix A...\n");
        setupMatrix(h_A, n, configs[i].type_A, configs[i].filename_A);
        print_matrix_stats(h_A, n, "A");

        printf("Setting up matrix B...\n");
        setupMatrix(h_B, n, configs[i].type_B, configs[i].filename_B);
        print_matrix_stats(h_B, n, "B");

        // Generate output filename
        char output_filename[256];
        snprintf(output_filename, sizeof(output_filename),
                "data/numerical_analysis_%s_n%d_tile%d.dat", configs[i].name, n, TILE_SIZE);

        printf("Running numerical analysis for %s...\n", configs[i].name);
        run_numerical_analysis(h_A, h_B, n, output_filename);
    }

    // Cleanup
    free(h_A);
    free(h_B);
}

// Generate comprehensive report from all test results
void generateReport(bool* enabled_sizes) {
    printf("\n=== Generating Numerical Analysis Report ===\n");

    FILE* summaryFile = fopen("data/numerical_analysis_summary.csv", "w");
    if (!summaryFile) {
        printf("ERROR: Could not create data/numerical_analysis_summary.csv\n");
        return;
    }

    fprintf(summaryFile, "test_name,size,avg_abs_error,max_rel_error,significant_errors,avg_condition_A,avg_condition_B\n");

    // TODO: Parse all generated .dat files and create summary statistics
    // For now, just write header and close
    printf("Report generation functionality to be implemented\n");
    printf("Summary will include analysis from all generated .dat files\n");

    fclose(summaryFile);
    printf("Report saved to: data/numerical_analysis_summary.csv\n");
}

// Main function for numerical analysis benchmarks - now refactored
void runNumericalAnalysisBenchmarks(bool* enabled_sizes) {
    printf("=== Starting Numerical Analysis of Tiled GEMM ===\n");

    // Run tests for each enabled size
    for (int size_idx = 0; size_idx < NUM_SIZES; size_idx++) {
        if (!enabled_sizes[size_idx]) continue;

        int n = SIZES[size_idx];
        printf("\n=== Testing matrix size %dx%d ===\n", n, n);

        // Test each matrix type using the working runMatrixTests function
        MatrixTestConfig configs[] = {
            {MATRIX_ODO_WELL_CONDITIONED, MATRIX_ODO_WELL_CONDITIONED, "wellcond", "Well-conditioned matrices", NULL, NULL, true},
            {MATRIX_ODO_ILL_CONDITIONED, MATRIX_ODO_ILL_CONDITIONED, "illcond", "Ill-conditioned matrices", NULL, NULL, true},
            {MATRIX_ZEROMEAN, MATRIX_ZEROMEAN, "zeromean", "Zero-mean distribution matrices", NULL, NULL, true},
            {MATRIX_UNIFORM_POSITIVE, MATRIX_UNIFORM_POSITIVE, "uniform_positive", "Uniform positive matrices", NULL, NULL, true},
            {MATRIX_RADEMACHER, MATRIX_RADEMACHER, "rademacher", "Rademacher distribution matrices", NULL, NULL, true}
        };

        int num_configs = sizeof(configs) / sizeof(configs[0]);
        runMatrixTests(n, configs, num_configs);
    }

    // Generate comprehensive report
    generateReport(enabled_sizes);

    printf("\nNumerical analysis complete!\n");
    printf("Individual test results: data/numerical_analysis_*.dat\n");
    printf("Summary report: data/numerical_analysis_summary.csv\n");
}


inline double gamma(int n, double u) {
    const double nu = n * u;
    return nu / (1.0 - nu);
}

inline int ceil_log2_int(int x) {
    int p = 0, v = x - 1;
    while (v > 0) { v >>= 1; ++p; }
    return p;
}

// Compute theoretical error bound factor based on kernel type and matrix size
float compute_beta_factor(KernelType kernel_type, bool single_pass, int n) {
    const double u = unit_roundoff_fp32(); // Use precision from config

    if (single_pass) {
        // For naive kernels: single-pass accumulation over n elements
        return (float)gamma(n, u);
    }

    // For tiled kernels: two-stage accumulation
    int num_tiles = (n + TILE_SIZE - 1) / TILE_SIZE;
    int tile_inner_k = TILE_SIZE; // Inner loop accumulation size

    // Stage 1: Accumulation within each tile (size TILE_SIZE)
    double beta_inner = gamma(tile_inner_k, u);

    // Stage 2: Accumulation across tiles
    bool pairwise = (kernel_type == KERNEL_TILED_PAIRWISE);
    double beta_outer;

    if (pairwise) {
        // Pairwise summation has logarithmic error growth
        beta_outer = gamma(ceil_log2_int(num_tiles), u);
    } else {
        // Standard summation has linear error growth
        beta_outer = gamma(num_tiles, u);
    }

    // Total error bound: inner + outer (conservative, cross-terms are O(u^2))
    return (float)(beta_inner + beta_outer);
}

// // Choose the right u: usually FP32 accumulation
// inline double beta_for_inner_k(int k, bool single_pass=true,
//                                int tile_b=0, int num_tiles=0, bool pairwise=false) {
//     const double u = unit_roundoff_fp32(); // change if accumulating in FP64, etc.
//     if (single_pass) return gamma(k, u);
//     // two-stage: micro-accumulate b, then reduce across t tiles
//     int b = tile_b, t = num_tiles;
//     if (b <= 0 || t <= 0) return gamma(k, u); // fallback
//     double beta_b = gamma(b, u);
//     double beta_t = pairwise ? gamma(ceil_log2_int(t), u) : gamma(t, u);
//     return beta_b + beta_t; // conservative; cross-terms are O(u^2)
// }

// Efficient multi-sample testing for specific matrix type and kernel
void run_multi_sample_analysis(MatrixType matrix_type, KernelType kernel_type, int n, int num_samples, const char* output_prefix) {
    printf("\n=== Multi-Sample Analysis ===\n");
    printf("Matrix Type: %d, Kernel: %d, Size: %dx%d, Samples: %d\n",
           (int)matrix_type, (int)kernel_type, n, n, num_samples);

    // Compute theoretical error bound factor
    // Use single_pass=false for tiled kernels, single_pass=true for naive kernel comparison
    bool single_pass = (kernel_type == KERNEL_NAIVE);
    float beta_factor = compute_beta_factor(kernel_type, single_pass, n);
    // Allocate device memory (reused across all samples)
    size_t size = n * n * sizeof(float);
    float *d_A, *d_B, *d_C_kernel;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C_kernel, size);

    // Allocate host memory for matrices and results
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C_kernel = (float*)malloc(size);
    float *h_C_reference = (float*)malloc(size);
    float *h_M_abs = (float*)malloc(size);

    // Statistics array for Frobenius norm only
    double *frobenius_errors = (double*)malloc(num_samples * sizeof(double));
    double *frobenius_M_error = (double*)malloc(num_samples * sizeof(double));
    double *normalized_errors = (double*)malloc(num_samples * sizeof(double));

    // Declare variables that might be accessed after goto
    FILE* fp = NULL;

    // Configure kernel launch parameters
    dim3 threadsPerBlock, numBlocks;
    compute_kernel_dimensions_dispatch(kernel_type, n, &threadsPerBlock, &numBlocks);

    printf("Running %d samples...\n", num_samples);

    // Run multiple samples
    for (int sample = 0; sample < num_samples; sample++) {
        if (sample % 10 == 0 && sample > 0) {
            printf("Completed %d/%d samples...\n", sample, num_samples);
        }

        // Generate new matrices for this sample using the specified matrix type
        // Use different seeds for each sample to ensure truly random matrices
        unsigned long long base_seed = (unsigned long long)time(NULL);
        generate_matrix_device_with_seed(d_A, n, matrix_type, base_seed + sample * 1000);
        generate_matrix_device_with_seed(d_B, n, matrix_type, base_seed + sample * 1000 + 1);

        // Copy matrices to host for CPU FP64 reference computation
        hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

        // Compute reference result using GPU FP64 (much faster than CPU)
        compute_C_reference_gpu_fp64(h_A, h_B, h_C_reference, n);

        // Launch the specified kernel using unified dispatch
        launch_kernel_by_type(kernel_type, d_A, d_B, d_C_kernel, n, numBlocks, threadsPerBlock);

        hipDeviceSynchronize();

        // Copy GPU result back to host for error computation
        hipMemcpy(h_C_kernel, d_C_kernel, size, hipMemcpyDeviceToHost);        // Compute Frobenius error for this sample
        double frobenius_error = 0.0;
        double frobenius_M = 0.0;

        // Copy matrices to host for CPU FP64 reference computation
        hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

        // Take absolute values on host arrays
        for (int i = 0; i < n * n; i++) {
            h_A[i] = fabsf(h_A[i]);
            h_B[i] = fabsf(h_B[i]);
        }

        // Compute reference result using GPU FP64 (much faster than CPU)
        compute_C_reference_gpu_fp64(h_A, h_B, h_M_abs, n);


        for (int i = 0; i < n * n; i++) {
            double diff_C = fabsf(h_C_kernel[i] - h_C_reference[i]);
            frobenius_error += diff_C * diff_C;
            double M_val = h_M_abs[i];
            frobenius_M += M_val * M_val;
        }

        frobenius_errors[sample] = sqrt(frobenius_error);
        frobenius_M_error[sample] = sqrt(frobenius_M);
        // Compute beta normalized error: empirical_error / (|A||B|)
        double theoretical_bound = frobenius_M_error[sample];
        normalized_errors[sample] = frobenius_errors[sample] / theoretical_bound;

    }

    printf("Completed all %d samples\n", num_samples);

    // Compute comprehensive statistics using utility function
    ArrayStats frob_stats;
    compute_array_statistics(frobenius_errors, num_samples, &frob_stats);

    ArrayStats beta_stats;
    compute_array_statistics(normalized_errors, num_samples, &beta_stats);

    // Print summary
    printf("\n=== Multi-Sample Analysis Results ===\n");
    printf("Matrix Type: %s, Kernel: %s, Size: %dx%d\n", matrixTypeToString(matrix_type), kernelTypeToString(kernel_type), n, n);
    printf("Number of samples: %d\n", num_samples);
    printf("\nFrobenius Error Statistics:\n");
    printf("  Average: %.3e\n", frob_stats.average);
    printf("  Std Dev: %.3e\n", frob_stats.std_dev);
    printf("  95th %%ile: %.3e\n", frob_stats.p95);
    printf("  Max: %.3e\n", frob_stats.maximum);
    printf("\nNormalized Error |C-C_ref|/(|A||B|) Statistics:\n");
    printf("  Average: %.3e\n", beta_stats.average);
    printf("  Std Dev: %.3e\n", beta_stats.std_dev);
    printf("  95th %%ile: %.3e\n", beta_stats.p95);
    printf("  Max: %.3e\n", beta_stats.maximum);
    printf("Theoretical error bound factor (beta): %.6e\n", beta_factor);
    printf("Average Error_beta/beta: %.6e\n", beta_stats.average/beta_factor);
    const double u32 = unit_roundoff_fp32();
    printf("Average Error_beta/u32: %.6e\n", beta_stats.average/u32);

    // Save summary results with metadata to file
    char filename[256];
    snprintf(filename, sizeof(filename), "data/%s_summary_n%d.csv", output_prefix, n);
    fp = fopen(filename, "w");
    if (fp) {
        // Write header with all metadata and statistics
        fprintf(fp, "matrix_type,kernel_type,matrix_size,num_samples,");
        fprintf(fp, "frob_avg,frob_std,frob_p95,frob_max,");
        fprintf(fp, "beta_avg,beta_std,beta_p95,beta_max,");
        fprintf(fp, "theoretical_beta,u32,beta_over_theoretical,beta_over_u32\n");

        // Write single row with all the summary data
        fprintf(fp, "%s,%s,%d,%d,",
                matrixTypeToString(matrix_type),
                kernelTypeToString(kernel_type),
                n,
                num_samples);
        fprintf(fp, "%.10e,%.10e,%.10e,%.10e,",
                frob_stats.average, frob_stats.std_dev, frob_stats.p95, frob_stats.maximum);
        fprintf(fp, "%.10e,%.10e,%.10e,%.10e,",
                beta_stats.average, beta_stats.std_dev, beta_stats.p95, beta_stats.maximum);
        fprintf(fp, "%.10e,%.10e,%.10e,%.10e\n",
                beta_factor, u32, beta_stats.average/beta_factor, beta_stats.average/u32);

        fclose(fp);
        printf("\nSummary results saved to: %s\n", filename);
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C_kernel);
    free(h_A); free(h_B); free(h_C_kernel); free(h_C_reference); free(h_M_abs);
    free(frobenius_errors); free(frobenius_M_error); free(normalized_errors);
}

#include "hip/hip_runtime.h"
#include "../include/utils.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "../include/gemms.cuh"
#include "../include/benchmark.h"  // For BLOCK_SIZE, TILE_SIZE constants

void fill_matrix(float *mat, int N) {
    for (int i = 0; i < N * N; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// Function to print CUDA device properties including theoretical performance
void printDevicePerformanceInfo() {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, 0);

    if (err != hipSuccess) {
        printf("Error getting device properties: %s\n", hipGetErrorString(err));
        return;
    }

    printf("\n===== DEVICE INFORMATION =====\n");
    printf("Device: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("Multiprocessor count: %d\n", prop.multiProcessorCount);

    // Calculate peak memory bandwidth (GB/s)
    // For RTX 4080, memory clock is ~21 GHz
    double memory_clock_rate = prop.memoryClockRate / 1000000.0; // Convert from kHz to GHz
    double bus_width = prop.memoryBusWidth;
    double peak_bandwidth = 2.0 * memory_clock_rate * (bus_width / 8); // GB/s

    // Calculate theoretical peak FLOPS for single precision
    // For RTX 4080, CUDA cores = 9728
    int cuda_cores = prop.multiProcessorCount * 128; // Approximate cores based on SM count
    double gpu_clock_ghz = prop.clockRate / 1000000.0; // Convert kHz to GHz
    double peak_gflops = 2.0 * cuda_cores * gpu_clock_ghz; // 2 ops per cycle with FMA

    printf("Memory clock rate (base estimate): %.1f GHz\n", memory_clock_rate);
    printf("Memory bus width: %d bits\n", prop.memoryBusWidth);
    printf("Peak memory bandwidth: %.2f GB/s\n", peak_bandwidth);
    printf("CUDA cores (estimate): %d\n", cuda_cores);
    printf("GPU clock: %.3f GHz\n", gpu_clock_ghz);
    printf("Peak performance (FP32): %.2f TFLOP/s\n", peak_gflops / 1000);
    printf("Arithmetic intensity ridge point: %.2f FLOP/byte\n", peak_gflops / peak_bandwidth);
    printf("\n");
}

// Compute reference result in FP64 on GPU using cuBLAS
void compute_C_reference_gpu_fp64(float *h_A, float *h_B, float *h_C_exact, int N) {
    //printf("Computing reference result in FP64 on GPU...\n");

    // Allocate GPU memory for FP64 computation
    size_t size_fp64 = N * N * sizeof(double);

    double *d_A_fp64, *d_B_fp64, *d_C_fp64;
    hipMalloc(&d_A_fp64, size_fp64);
    hipMalloc(&d_B_fp64, size_fp64);
    hipMalloc(&d_C_fp64, size_fp64);

    // Allocate host memory for FP64 data
    double *h_A_fp64 = (double*)malloc(size_fp64);
    double *h_B_fp64 = (double*)malloc(size_fp64);
    double *h_C_fp64 = (double*)malloc(size_fp64);

    // Convert FP32 to FP64 on CPU (fast conversion, GPU GEMM still dominates)
    for (int i = 0; i < N * N; i++) {
        h_A_fp64[i] = (double)h_A[i];
        h_B_fp64[i] = (double)h_B[i];
    }

    // Copy FP64 data to GPU
    hipMemcpy(d_A_fp64, h_A_fp64, size_fp64, hipMemcpyHostToDevice);
    hipMemcpy(d_B_fp64, h_B_fp64, size_fp64, hipMemcpyHostToDevice);

    // Create cuBLAS handle for FP64
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform FP64 GEMM on GPU
    const double alpha = 1.0, beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                &alpha, d_B_fp64, N, d_A_fp64, N, &beta, d_C_fp64, N);

    // Copy result back to host
    hipMemcpy(h_C_fp64, d_C_fp64, size_fp64, hipMemcpyDeviceToHost);

    // Convert back to FP32 for compatibility
    for (int i = 0; i < N * N; i++) {
        h_C_exact[i] = (float)h_C_fp64[i];
    }

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A_fp64); hipFree(d_B_fp64); hipFree(d_C_fp64);
    free(h_A_fp64); free(h_B_fp64); free(h_C_fp64);
}

// Compute reference result in FP64 on CPU
void compute_C_reference(float *A, float *B, float *C_exact, int N) {
    //printf("Computing reference result in FP64 on CPU...\n");

    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            // Calculate reference in double precision
            double sum = 0.0;
            for (int k = 0; k < N; ++k) {
                sum += (double)A[row * N + k] * (double)B[k * N + col];
            }
            // Store result as float
            C_exact[row * N + col] = (float)sum;
        }
    }
}

void verify_result(float *A, float *B, float *C, int N) {
    // Use more appropriate epsilon for float comparison
    float eps = 1e-6;
    float max_rel_error = 0.0f;
    float sum_rel_error = 0.0f;
    int error_count = 0;

    // Allocate memory for CPU FP64 reference
    float *C_exact = (float*)malloc(N * N * sizeof(float));
    compute_C_reference(A, B, C_exact, N);

    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            int idx = row * N + col;

            // Compare GPU float result with CPU FP64 reference
            double abs_error = fabs((double)C[idx] - (double)C_exact[idx]);
            double rel_error = abs_error / (fabs((double)C_exact[idx]) > 1e-10 ? fabs((double)C_exact[idx]) : 1e-10);

            // Record statistics
            max_rel_error = fmax(max_rel_error, (float)rel_error);
            sum_rel_error += (float)rel_error;

            // Check error threshold
            if (abs_error > (double)eps) {
                error_count++;
                if (error_count <= 5) { // Limit output to first 5 errors
                    printf("Mismatch at (%d, %d): GPU = %f, CPU_FP64 = %f, Rel Error = %e\n",
                          row, col, C[idx], C_exact[idx], rel_error);
                }
            }
        }
    }

    printf("Max relative error: %e\n", max_rel_error);
    printf("Average relative error: %e\n", sum_rel_error / (N * N));
    printf("Number of elements with error > %e: %d (%.2f%%)\n",
           eps, error_count, 100.0f * error_count / (N * N));

    if (error_count == 0)
        printf("Result verified: correct within epsilon %e.\n", eps);

    // Clean up
    free(C_exact);
}
void check_occupancy() {
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    // Check occupancy for different configurations
    int maxActiveBlocks16, maxActiveBlocks32;

    // For TILE_SIZE=16 (16×16 = 256 threads per block)
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks16,
                                                  (void(*)(float*, float*, float*, int))matmul_tiled, 256, 2048);

    // For TILE_SIZE=32 (32×32 = 1024 threads per block)
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks32,
                                                  (void(*)(float*, float*, float*, int))matmul_tiled, 1024, 8192);

    printf("=== OCCUPANCY ANALYSIS ===\n");
    printf("GPU: %s, SMs: %d\n", prop.name, prop.multiProcessorCount);
    printf("Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("Max blocks per SM: %d\n", prop.maxBlocksPerMultiProcessor);

    printf("\nTILE_SIZE=16 (256 threads/block, 2KB shared mem):\n");
    printf("  Max active blocks per SM: %d\n", maxActiveBlocks16);
    printf("  Threads per SM: %d (%.1f%% occupancy)\n",
           maxActiveBlocks16 * 256,
           (maxActiveBlocks16 * 256.0f / prop.maxThreadsPerMultiProcessor) * 100);

    printf("\nTILE_SIZE=32 (1024 threads/block, 8KB shared mem):\n");
    printf("  Max active blocks per SM: %d\n", maxActiveBlocks32);
    printf("  Threads per SM: %d (%.1f%% occupancy)\n",
           maxActiveBlocks32 * 1024,
           (maxActiveBlocks32 * 1024.0f / prop.maxThreadsPerMultiProcessor) * 100);
}

void printCacheInfo() {
    printf("\n===== DETAILED CACHE ANALYSIS =====\n");

    int value;
    hipError_t err;

    // L1 Global Cache
    err = hipDeviceGetAttribute(&value, hipDeviceAttributeGlobalL1CacheSupported, 0);
    printf("Global L1 Cache Supported: %s\n",
           (err == hipSuccess) ? (value ? "YES" : "NO") : "UNKNOWN");

    // L1 Local Cache
    err = hipDeviceGetAttribute(&value, hipDeviceAttributeLocalL1CacheSupported, 0);
    printf("Local L1 Cache Supported: %s\n",
           (err == hipSuccess) ? (value ? "YES" : "NO") : "UNKNOWN");

    // L2 Cache Size
    err = hipDeviceGetAttribute(&value, hipDeviceAttributeL2CacheSize, 0);
    if (err == hipSuccess) {
        printf("L2 Cache Size: %d bytes (%.2f MB)\n", value, value / (1024.0 * 1024.0));
    }

    // Cache configuration
    hipFuncCache_t cacheConfig;
    err = hipDeviceGetCacheConfig(&cacheConfig);
    if (err == hipSuccess) {
        printf("Current cache preference: ");
        switch(cacheConfig) {
            case hipFuncCachePreferNone: printf("No preference\n"); break;
            case hipFuncCachePreferShared: printf("Prefer shared memory\n"); break;
            case hipFuncCachePreferL1: printf("Prefer L1 cache\n"); break;
            case hipFuncCachePreferEqual: printf("Equal L1/shared\n"); break;
            default: printf("Unknown\n"); break;
        }
    }

    // Memory transaction size (cache line related)
    printf("\nMemory Transaction Analysis:\n");
    printf("Expected cache line size: 128 bytes (32 floats)\n");
    printf("Your warp accesses: 32 consecutive floats\n");
    printf("Theoretical coalescing: PERFECT (1:1 ratio expected)\n");

    printf("=====================================\n\n");
}

// Kernel resource assessment function
void assess_kernel_resources(KernelType kernel_type, int n) {
    printf("\n=== Kernel Resource Assessment ===\n");
    printf("Kernel: %s, Matrix Size: %dx%d\n", kernelTypeToString(kernel_type), n, n);

    hipFuncAttributes attr = {0}; // Initialize to zero
    hipError_t err = hipErrorInvalidDeviceFunction; // Default to error state

    // Note: For CUTLASS kernels, we can't easily get function pointers since they're template-generated
    // This function will focus on the kernels we can assess directly
    switch(kernel_type) {
        case KERNEL_NAIVE:
            err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(matmul_naive));
            if (err != hipSuccess) {
                printf("  Error getting naive kernel attributes: %s\n", hipGetErrorString(err));
            }
            break;

        case KERNEL_TILED:
            err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(matmul_tiled));
            if (err != hipSuccess) {
                printf("  Error getting tiled kernel attributes: %s\n", hipGetErrorString(err));
            }
            break;

        case KERNEL_TILED_OPT:
            err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(matmul_tiled_opt));
            if (err != hipSuccess) {
                printf("  Error getting tiled optimized kernel attributes: %s\n", hipGetErrorString(err));
            }
            break;

        case KERNEL_TILED_PAIRWISE:
            err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(matmul_tiled_pairwise));
            if (err != hipSuccess) {
                printf("  Error getting tiled pairwise kernel attributes: %s\n", hipGetErrorString(err));
            }
            break;

        case KERNEL_TILED_RECT:
            err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(matmul_tiled_rectangular));
            if (err != hipSuccess) {
                printf("  Error getting tiled rectangular kernel attributes: %s\n", hipGetErrorString(err));
            }
            break;

        case KERNEL_CUTLASS_SPLITK_FLAT:
            printf("  Note: CUTLASS Split-K Flat uses template-generated kernels\n");
            printf("  Resource usage depends on CUTLASS template instantiation\n");
            printf("  Cannot retrieve detailed attributes for template kernels\n");
            break;

        case KERNEL_CUTLASS_SPLITK_PAIRWISE:
            printf("  Note: CUTLASS Split-K Pairwise uses template-generated kernels\n");
            printf("  Resource usage depends on CUTLASS template instantiation\n");
            printf("  Cannot retrieve detailed attributes for template kernels\n");
            break;

        case KERNEL_CUBLAS:
        case KERNEL_CUBLAS_TENSOR:
            printf("  Note: cuBLAS kernels are proprietary and cannot be assessed\n");
            break;

        case KERNEL_CUTLASS:
        case KERNEL_CUTLASS_TENSOR:
            printf("  Note: CUTLASS kernels use template-generated code\n");
            printf("  Resource usage depends on CUTLASS template instantiation\n");
            printf("  Cannot retrieve detailed attributes for template kernels\n");
            break;

        case KERNEL_TILED_MIXPREC:
            printf("  Note: Mixed precision kernel uses compile-time type configuration\n");
            printf("  Current types: COMPUTE_TYPE=%s, ACCUMULATE_TYPE=%s\n",
                   // You might want to add type name macros to your config
                   "configured at build time", "configured at build time");
            // Can't get attributes for template kernel easily
            break;

        default:
            printf("  Unknown kernel type\n");
            break;
    }

    // Only show kernel attributes if we successfully retrieved them
    if (err == hipSuccess) {
        printf("  Kernel Resource Details:\n");
        printf("    Registers per thread: %d\n", attr.numRegs);
        printf("    Shared memory per block: %zu bytes\n", attr.sharedSizeBytes);
        printf("    Max threads per block: %d\n", attr.maxThreadsPerBlock);
        printf("    Constant memory: %zu bytes\n", attr.constSizeBytes);
        printf("    Local memory per thread: %zu bytes\n", attr.localSizeBytes);
    }

    // Calculate theoretical occupancy for this kernel
    dim3 threadsPerBlock, numBlocks;
    compute_kernel_dimensions_dispatch(kernel_type, n, &threadsPerBlock, &numBlocks);

    printf("  Launch configuration:\n");
    printf("    Threads per block: %d x %d = %d\n",
           threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.x * threadsPerBlock.y);
    printf("    Grid size: %d x %d = %d blocks\n",
           numBlocks.x, numBlocks.y, numBlocks.x * numBlocks.y);
    printf("    Total threads: %d\n",
           (numBlocks.x * numBlocks.y) * (threadsPerBlock.x * threadsPerBlock.y));

    // Calculate and report occupancy for native CUDA kernels
    if (err == hipSuccess) {
        int device;
        hipGetDevice(&device);
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);

        int maxActiveBlocks = 0;
        int threadsPerBlockTotal = threadsPerBlock.x * threadsPerBlock.y;
        size_t sharedMemPerBlock = attr.sharedSizeBytes;

        // Calculate occupancy based on the specific kernel
        void* kernel_ptr = nullptr;
        switch(kernel_type) {
            case KERNEL_NAIVE:
                kernel_ptr = (void*)matmul_naive;
                break;
            case KERNEL_TILED:
                kernel_ptr = (void*)matmul_tiled;
                break;
            case KERNEL_TILED_OPT:
                kernel_ptr = (void*)matmul_tiled_opt;
                break;
            case KERNEL_TILED_PAIRWISE:
                kernel_ptr = (void*)matmul_tiled_pairwise;
                break;
            case KERNEL_TILED_RECT:
                kernel_ptr = (void*)matmul_tiled_rectangular;
                break;
            default:
                kernel_ptr = nullptr;
                break;
        }

        if (kernel_ptr != nullptr) {
            hipError_t occ_err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &maxActiveBlocks, kernel_ptr, threadsPerBlockTotal, sharedMemPerBlock);

            if (occ_err == hipSuccess) {
                int activeThreadsPerSM = maxActiveBlocks * threadsPerBlockTotal;
                double occupancy_percent = (activeThreadsPerSM * 100.0) / prop.maxThreadsPerMultiProcessor;

                printf("  Occupancy Analysis:\n");
                printf("    Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
                printf("    Max blocks per SM: %d\n", prop.maxBlocksPerMultiProcessor);
                printf("    Active blocks per SM: %d\n", maxActiveBlocks);
                printf("    Active threads per SM: %d\n", activeThreadsPerSM);
                printf("    Theoretical occupancy: %.1f%%\n", occupancy_percent);

                // Calculate limiting factors
                int max_blocks_by_threads = prop.maxThreadsPerMultiProcessor / threadsPerBlockTotal;
                int max_blocks_by_sm_limit = prop.maxBlocksPerMultiProcessor;

                printf("    Limiting factors:\n");
                printf("      Max blocks by thread limit: %d\n", max_blocks_by_threads);
                printf("      Max blocks by SM limit: %d\n", max_blocks_by_sm_limit);
                if (sharedMemPerBlock > 0) {
                    int max_blocks_by_shared_mem = prop.sharedMemPerMultiprocessor / sharedMemPerBlock;
                    printf("      Max blocks by shared memory: %d\n", max_blocks_by_shared_mem);
                }
            } else {
                printf("  Occupancy Analysis: Error calculating occupancy: %s\n", hipGetErrorString(occ_err));
            }
        }
    }

    printf("===================================\n\n");
}

// Comparison function for qsort (for percentile calculation)
static int compare_doubles(const void *a, const void *b) {
    double arg1 = *(const double*)a;
    double arg2 = *(const double*)b;

    if (arg1 < arg2) return -1;
    if (arg1 > arg2) return 1;
    return 0;
}

// Compute comprehensive statistics for an array of doubles
void compute_array_statistics(const double* array, int size, ArrayStats* stats) {
    if (size <= 0 || array == NULL || stats == NULL) {
        return;
    }

    // Copy array for sorting (needed for percentile)
    double* sorted_array = (double*)malloc(size * sizeof(double));
    memcpy(sorted_array, array, size * sizeof(double));

    // Calculate average and find min/max
    double sum = 0.0;
    stats->minimum = array[0];
    stats->maximum = array[0];

    for (int i = 0; i < size; i++) {
        sum += array[i];
        if (array[i] < stats->minimum) stats->minimum = array[i];
        if (array[i] > stats->maximum) stats->maximum = array[i];
    }
    stats->average = sum / size;

    // Calculate standard deviation
    double variance_sum = 0.0;
    for (int i = 0; i < size; i++) {
        double diff = array[i] - stats->average;
        variance_sum += diff * diff;
    }
    stats->std_dev = sqrt(variance_sum / size);

    // Sort array and calculate percentiles
    qsort(sorted_array, size, sizeof(double), compare_doubles);

    // Calculate 10th percentile
    double percentile_index_10 = 0.10 * (size - 1);
    int lower_index_10 = (int)floor(percentile_index_10);
    int upper_index_10 = (int)ceil(percentile_index_10);

    if (lower_index_10 == upper_index_10) {
        stats->p10 = sorted_array[lower_index_10];
    } else {
        // Linear interpolation between the two closest values
        double weight_10 = percentile_index_10 - lower_index_10;
        stats->p10 = sorted_array[lower_index_10] * (1.0 - weight_10) + sorted_array[upper_index_10] * weight_10;
    }

    // Calculate 95th percentile
    double percentile_index = 0.95 * (size - 1);
    int lower_index = (int)floor(percentile_index);
    int upper_index = (int)ceil(percentile_index);

    if (lower_index == upper_index) {
        stats->p95 = sorted_array[lower_index];
    } else {
        // Linear interpolation between the two closest values
        double weight = percentile_index - lower_index;
        stats->p95 = sorted_array[lower_index] * (1.0 - weight) + sorted_array[upper_index] * weight;
    }

    free(sorted_array);
}

// Unified kernel dispatch function that both benchmark and error analysis can use
// Function to map kernel name to KernelType
KernelType getKernelTypeFromName(const char* name) {
    if (strcmp(name, "naive") == 0) return KERNEL_NAIVE;
    if (strcmp(name, "tiled") == 0) return KERNEL_TILED;
    if (strcmp(name, "tiled_opt") == 0) return KERNEL_TILED_OPT;
    if (strcmp(name, "tiled_pairwise") == 0) return KERNEL_TILED_PAIRWISE;
    if (strcmp(name, "tiled_rect") == 0) return KERNEL_TILED_RECT;
    if (strcmp(name, "tiled_mixprec") == 0) return KERNEL_TILED_MIXPREC;
    if (strcmp(name, "tiled_pairwise_mixprec") == 0) return KERNEL_TILED_PAIRWISE_MIXPREC;
    if (strcmp(name, "cublas") == 0) return KERNEL_CUBLAS;
    if (strcmp(name, "cublas_tensor") == 0) return KERNEL_CUBLAS_TENSOR;
    if (strcmp(name, "cutlass") == 0) return KERNEL_CUTLASS;
    if (strcmp(name, "cutlass_tensor") == 0) return KERNEL_CUTLASS_TENSOR;
    if (strcmp(name, "cutlass_splitk_flat") == 0) return KERNEL_CUTLASS_SPLITK_FLAT;
    if (strcmp(name, "cutlass_splitk_pairwise") == 0) return KERNEL_CUTLASS_SPLITK_PAIRWISE;
    return static_cast<KernelType>(-1); // Return invalid value for unknown names
}

// Function to map matrix type name to MatrixType
MatrixType getMatrixTypeFromName(const char* name) {
    if (strcmp(name, "wellcond") == 0) return MATRIX_ODO_WELL_CONDITIONED;
    if (strcmp(name, "illcond") == 0) return MATRIX_ODO_ILL_CONDITIONED;
    if (strcmp(name, "zeromean") == 0) return MATRIX_ZEROMEAN;
    if (strcmp(name, "uniform_positive") == 0) return MATRIX_UNIFORM_POSITIVE;
    if (strcmp(name, "2powers") == 0) return MATRIX_SCALED_2POWERS;
    if (strcmp(name, "rademacher") == 0) return MATRIX_RADEMACHER;
    if (strcmp(name, "sanity") == 0) return MATRIX_SANITY;
    if (strcmp(name, "lognormal") == 0) return MATRIX_LOGNORMAL;
    if (strcmp(name, "file") == 0) return MATRIX_FROM_FILE;
    return static_cast<MatrixType>(-1); // Return invalid value for unknown names
}

// Reverse conversion functions: enum to string
const char* kernelTypeToString(KernelType kernel_type) {
    switch(kernel_type) {
        case KERNEL_NAIVE: return "naive";
        case KERNEL_TILED: return "tiled";
        case KERNEL_TILED_OPT: return "tiled_opt";
        case KERNEL_TILED_PAIRWISE: return "tiled_pairwise";
        case KERNEL_TILED_RECT: return "tiled_rect";
        case KERNEL_TILED_MIXPREC: return "tiled_mixprec";
        case KERNEL_TILED_PAIRWISE_MIXPREC: return "tiled_pairwise_mixprec";  // Add this
        case KERNEL_CUBLAS: return "cublas";
        case KERNEL_CUBLAS_TENSOR: return "cublas_tensor";
        case KERNEL_CUTLASS: return "cutlass";
        case KERNEL_CUTLASS_TENSOR: return "cutlass_tensor";
        case KERNEL_CUTLASS_SPLITK_FLAT: return "cutlass_splitk_flat";
        case KERNEL_CUTLASS_SPLITK_PAIRWISE: return "cutlass_splitk_pairwise";
        default: return "unknown";
    }
}

const char* matrixTypeToString(MatrixType matrix_type) {
    switch(matrix_type) {
        case MATRIX_ODO_WELL_CONDITIONED: return "wellcond";
        case MATRIX_ODO_ILL_CONDITIONED: return "illcond";
        case MATRIX_ZEROMEAN: return "zeromean";
        case MATRIX_UNIFORM_POSITIVE: return "uniform_positive";
        case MATRIX_SCALED_2POWERS: return "2powers";
        case MATRIX_RADEMACHER: return "rademacher";
        case MATRIX_SANITY: return "sanity";
        case MATRIX_LOGNORMAL: return "lognormal";
        case MATRIX_FROM_FILE: return "file";
        default: return "unknown";
    }
}

// Optimized kernel dispatch using function pointer table
typedef void (*KernelFunc)(float*, float*, float*, int, dim3, dim3);

static KernelFunc kernel_function_table[] = {
    launch_naive,                           // KERNEL_NAIVE
    launch_tiled,                          // KERNEL_TILED
    launch_tiled_opt,                      // KERNEL_TILED_OPT
    launch_tiled_pairwise,                 // KERNEL_TILED_PAIRWISE
    launch_tiled_rect,                     // KERNEL_TILED_RECT
    launch_tiled_mixprec,           // KERNEL_TILED_MIXPREC
    launch_tiled_pairwise_mixprec,  // KERNEL_TILED_PAIRWISE_MIXPREC - Add this
    launch_cublas,                         // KERNEL_CUBLAS
    launch_cublas_tensor,                  // KERNEL_CUBLAS_TENSOR
    launch_cutlass,                        // KERNEL_CUTLASS
    launch_cutlass_tensor,                 // KERNEL_CUTLASS_TENSOR
    launch_cutlass_splitk_flat,            // KERNEL_CUTLASS_SPLITK_FLAT
    launch_cutlass_splitk_pairwise         // KERNEL_CUTLASS_SPLITK_PAIRWISE
};

void launch_kernel_by_type(KernelType kernel_type, float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Bounds check for safety
    if (kernel_type < 0 || kernel_type >= sizeof(kernel_function_table)/sizeof(kernel_function_table[0])) {
        printf("ERROR: Invalid kernel type %d\n", (int)kernel_type);
        return;
    }

    // Direct function pointer call - zero overhead dispatch!
    kernel_function_table[kernel_type](d_A, d_B, d_C, n, blocks, threads);
}

// Template-based kernel dimension computation for compile-time efficiency
template<KernelType kernel_type>
void compute_kernel_dimensions_template(int n, dim3* threadsPerBlock, dim3* numBlocks);

// Template specializations for each kernel type
template<>
void compute_kernel_dimensions_template<KERNEL_NAIVE>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    *threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    *numBlocks = dim3((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

template<>
void compute_kernel_dimensions_template<KERNEL_TILED>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    *threadsPerBlock = dim3(TILE_SIZE, TILE_SIZE);
    *numBlocks = dim3((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);
}

template<>
void compute_kernel_dimensions_template<KERNEL_TILED_RECT>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    *threadsPerBlock = dim3(BLOCK_N, BLOCK_M);
    *numBlocks = dim3((n + TILE_N - 1) / TILE_N, (n + TILE_M - 1) / TILE_M);
}

template<>
void compute_kernel_dimensions_template<KERNEL_CUBLAS>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    *threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    *numBlocks = dim3((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

template<>
void compute_kernel_dimensions_template<KERNEL_CUBLAS_TENSOR>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    *threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    *numBlocks = dim3((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

template<>
void compute_kernel_dimensions_template<KERNEL_CUTLASS>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    *threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    *numBlocks = dim3((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

template<>
void compute_kernel_dimensions_template<KERNEL_CUTLASS_TENSOR>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    *threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    *numBlocks = dim3((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

template<>
void compute_kernel_dimensions_template<KERNEL_CUTLASS_SPLITK_FLAT>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    // Use standard 2D configuration for split-K flat implementation
    *threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    *numBlocks = dim3((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

template<>
void compute_kernel_dimensions_template<KERNEL_CUTLASS_SPLITK_PAIRWISE>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    // Use standard 2D configuration for split-K pairwise implementation
    *threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    *numBlocks = dim3((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

template<>
void compute_kernel_dimensions_template<KERNEL_HELPER_1D>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    // Compute total elements from matrix dimension (like other kernels)
    const int total_elements = n * n;

    *threadsPerBlock = dim3(256);
    *numBlocks = dim3((total_elements + 256 - 1) / 256);
}

// Add template specialization for mixed precision kernel
template<>
void compute_kernel_dimensions_template<KERNEL_TILED_MIXPREC>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    *threadsPerBlock = dim3(TILE_SIZE, TILE_SIZE);
    *numBlocks = dim3((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);
}

template<>
void compute_kernel_dimensions_template<KERNEL_TILED_PAIRWISE_MIXPREC>(int n, dim3* threadsPerBlock, dim3* numBlocks) {
    *threadsPerBlock = dim3(TILE_SIZE, TILE_SIZE);
    *numBlocks = dim3((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);
}

// Runtime dispatch function that calls the appropriate template specialization
void compute_kernel_dimensions_dispatch(KernelType kernel_type, int n, dim3* threadsPerBlock, dim3* numBlocks) {
    switch(kernel_type) {
        case KERNEL_NAIVE:
            compute_kernel_dimensions_template<KERNEL_NAIVE>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_TILED:
            compute_kernel_dimensions_template<KERNEL_TILED>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_TILED_RECT:
            compute_kernel_dimensions_template<KERNEL_TILED_RECT>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_CUBLAS:
            compute_kernel_dimensions_template<KERNEL_CUBLAS>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_CUBLAS_TENSOR:
            compute_kernel_dimensions_template<KERNEL_CUBLAS_TENSOR>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_CUTLASS:
            compute_kernel_dimensions_template<KERNEL_CUTLASS>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_CUTLASS_TENSOR:
            compute_kernel_dimensions_template<KERNEL_CUTLASS_TENSOR>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_CUTLASS_SPLITK_FLAT:
            compute_kernel_dimensions_template<KERNEL_CUTLASS_SPLITK_FLAT>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_CUTLASS_SPLITK_PAIRWISE:
            compute_kernel_dimensions_template<KERNEL_CUTLASS_SPLITK_PAIRWISE>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_HELPER_1D:
            compute_kernel_dimensions_template<KERNEL_HELPER_1D>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_TILED_MIXPREC:
            compute_kernel_dimensions_template<KERNEL_TILED_MIXPREC>(n, threadsPerBlock, numBlocks);
            break;
        case KERNEL_TILED_PAIRWISE_MIXPREC:
            compute_kernel_dimensions_template<KERNEL_TILED_PAIRWISE_MIXPREC>(n, threadsPerBlock, numBlocks);
            break;
        default:
            // Default fallback
            *threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
            *numBlocks = dim3((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
            break;
    }
}

// 1D kernel dimension dispatch function for helper kernels
void compute_kernel_dimensions_dispatch_1D(int total_elements, int* threadsPerBlock, int* numBlocks) {
    // Standard 1D configuration for element-wise operations
    *threadsPerBlock = 256;  // Optimal block size for most GPUs
    *numBlocks = (total_elements + 256 - 1) / 256;
}
// Efficient string-based wrapper that uses templates internally
void compute_dimensions(const char* kernel_name, int n, dim3* threadsPerBlock, dim3* numBlocks) {
    KernelType kernel_type = getKernelTypeFromName(kernel_name);
    compute_kernel_dimensions_dispatch(kernel_type, n, threadsPerBlock, numBlocks);
}
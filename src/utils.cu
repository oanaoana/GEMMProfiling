#include "hip/hip_runtime.h"
#include "../include/utils.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "../include/gemms.cuh"
#include "../include/benchmark.h"  // For BLOCK_SIZE, TILE_SIZE constants

void fill_matrix(float *mat, int N) {
    for (int i = 0; i < N * N; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// Function to print CUDA device properties including theoretical performance
void printDevicePerformanceInfo() {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, 0);

    if (err != hipSuccess) {
        printf("Error getting device properties: %s\n", hipGetErrorString(err));
        return;
    }

    printf("\n===== DEVICE INFORMATION =====\n");
    printf("Device: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("Multiprocessor count: %d\n", prop.multiProcessorCount);

    // Calculate peak memory bandwidth (GB/s)
    // For RTX 4080, memory clock is ~21 GHz
    double memory_clock_rate = prop.memoryClockRate / 1000000.0; // Convert from kHz to GHz
    double bus_width = prop.memoryBusWidth;
    double peak_bandwidth = 2.0 * memory_clock_rate * (bus_width / 8); // GB/s

    // Calculate theoretical peak FLOPS for single precision
    // For RTX 4080, CUDA cores = 9728
    int cuda_cores = prop.multiProcessorCount * 128; // Approximate cores based on SM count
    double gpu_clock_ghz = prop.clockRate / 1000000.0; // Convert kHz to GHz
    double peak_gflops = 2.0 * cuda_cores * gpu_clock_ghz; // 2 ops per cycle with FMA

    printf("Memory clock rate (base estimate): %.1f GHz\n", memory_clock_rate);
    printf("Memory bus width: %d bits\n", prop.memoryBusWidth);
    printf("Peak memory bandwidth: %.2f GB/s\n", peak_bandwidth);
    printf("CUDA cores (estimate): %d\n", cuda_cores);
    printf("GPU clock: %.3f GHz\n", gpu_clock_ghz);
    printf("Peak performance (FP32): %.2f TFLOP/s\n", peak_gflops / 1000);
    printf("Arithmetic intensity ridge point: %.2f FLOP/byte\n", peak_gflops / peak_bandwidth);
    printf("\n");
}

// Compute reference result in FP64 on GPU using cuBLAS
void compute_C_reference_gpu_fp64(float *h_A, float *h_B, float *h_C_exact, int N) {
    //printf("Computing reference result in FP64 on GPU...\n");

    // Allocate GPU memory for FP64 computation
    size_t size_fp64 = N * N * sizeof(double);

    double *d_A_fp64, *d_B_fp64, *d_C_fp64;
    hipMalloc(&d_A_fp64, size_fp64);
    hipMalloc(&d_B_fp64, size_fp64);
    hipMalloc(&d_C_fp64, size_fp64);

    // Allocate host memory for FP64 data
    double *h_A_fp64 = (double*)malloc(size_fp64);
    double *h_B_fp64 = (double*)malloc(size_fp64);
    double *h_C_fp64 = (double*)malloc(size_fp64);

    // Convert FP32 to FP64 on CPU (fast conversion, GPU GEMM still dominates)
    for (int i = 0; i < N * N; i++) {
        h_A_fp64[i] = (double)h_A[i];
        h_B_fp64[i] = (double)h_B[i];
    }

    // Copy FP64 data to GPU
    hipMemcpy(d_A_fp64, h_A_fp64, size_fp64, hipMemcpyHostToDevice);
    hipMemcpy(d_B_fp64, h_B_fp64, size_fp64, hipMemcpyHostToDevice);

    // Create cuBLAS handle for FP64
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform FP64 GEMM on GPU
    const double alpha = 1.0, beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                &alpha, d_B_fp64, N, d_A_fp64, N, &beta, d_C_fp64, N);

    // Copy result back to host
    hipMemcpy(h_C_fp64, d_C_fp64, size_fp64, hipMemcpyDeviceToHost);

    // Convert back to FP32 for compatibility
    for (int i = 0; i < N * N; i++) {
        h_C_exact[i] = (float)h_C_fp64[i];
    }

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A_fp64); hipFree(d_B_fp64); hipFree(d_C_fp64);
    free(h_A_fp64); free(h_B_fp64); free(h_C_fp64);
}

// Compute reference result in FP64 on CPU
void compute_C_reference(float *A, float *B, float *C_exact, int N) {
    //printf("Computing reference result in FP64 on CPU...\n");

    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            // Calculate reference in double precision
            double sum = 0.0;
            for (int k = 0; k < N; ++k) {
                sum += (double)A[row * N + k] * (double)B[k * N + col];
            }
            // Store result as float
            C_exact[row * N + col] = (float)sum;
        }
    }
}

void verify_result(float *A, float *B, float *C, int N) {
    // Use more appropriate epsilon for float comparison
    float eps = 1e-6;
    float max_rel_error = 0.0f;
    float sum_rel_error = 0.0f;
    int error_count = 0;

    // Allocate memory for CPU FP64 reference
    float *C_exact = (float*)malloc(N * N * sizeof(float));
    compute_C_reference(A, B, C_exact, N);

    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            int idx = row * N + col;

            // Compare GPU float result with CPU FP64 reference
            double abs_error = fabs((double)C[idx] - (double)C_exact[idx]);
            double rel_error = abs_error / (fabs((double)C_exact[idx]) > 1e-10 ? fabs((double)C_exact[idx]) : 1e-10);

            // Record statistics
            max_rel_error = fmax(max_rel_error, (float)rel_error);
            sum_rel_error += (float)rel_error;

            // Check error threshold
            if (abs_error > (double)eps) {
                error_count++;
                if (error_count <= 5) { // Limit output to first 5 errors
                    printf("Mismatch at (%d, %d): GPU = %f, CPU_FP64 = %f, Rel Error = %e\n",
                          row, col, C[idx], C_exact[idx], rel_error);
                }
            }
        }
    }

    printf("Max relative error: %e\n", max_rel_error);
    printf("Average relative error: %e\n", sum_rel_error / (N * N));
    printf("Number of elements with error > %e: %d (%.2f%%)\n",
           eps, error_count, 100.0f * error_count / (N * N));

    if (error_count == 0)
        printf("Result verified: correct within epsilon %e.\n", eps);

    // Clean up
    free(C_exact);
}
void check_occupancy() {
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    // Check occupancy for different configurations
    int maxActiveBlocks16, maxActiveBlocks32;

    // For TILE_SIZE=16 (16×16 = 256 threads per block)
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks16,
                                                  (void(*)(float*, float*, float*, int))matmul_tiled, 256, 2048);

    // For TILE_SIZE=32 (32×32 = 1024 threads per block)
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks32,
                                                  (void(*)(float*, float*, float*, int))matmul_tiled, 1024, 8192);

    printf("=== OCCUPANCY ANALYSIS ===\n");
    printf("GPU: %s, SMs: %d\n", prop.name, prop.multiProcessorCount);
    printf("Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("Max blocks per SM: %d\n", prop.maxBlocksPerMultiProcessor);

    printf("\nTILE_SIZE=16 (256 threads/block, 2KB shared mem):\n");
    printf("  Max active blocks per SM: %d\n", maxActiveBlocks16);
    printf("  Threads per SM: %d (%.1f%% occupancy)\n",
           maxActiveBlocks16 * 256,
           (maxActiveBlocks16 * 256.0f / prop.maxThreadsPerMultiProcessor) * 100);

    printf("\nTILE_SIZE=32 (1024 threads/block, 8KB shared mem):\n");
    printf("  Max active blocks per SM: %d\n", maxActiveBlocks32);
    printf("  Threads per SM: %d (%.1f%% occupancy)\n",
           maxActiveBlocks32 * 1024,
           (maxActiveBlocks32 * 1024.0f / prop.maxThreadsPerMultiProcessor) * 100);
}

void printCacheInfo() {
    printf("\n===== DETAILED CACHE ANALYSIS =====\n");

    int value;
    hipError_t err;

    // L1 Global Cache
    err = hipDeviceGetAttribute(&value, hipDeviceAttributeGlobalL1CacheSupported, 0);
    printf("Global L1 Cache Supported: %s\n",
           (err == hipSuccess) ? (value ? "YES" : "NO") : "UNKNOWN");

    // L1 Local Cache
    err = hipDeviceGetAttribute(&value, hipDeviceAttributeLocalL1CacheSupported, 0);
    printf("Local L1 Cache Supported: %s\n",
           (err == hipSuccess) ? (value ? "YES" : "NO") : "UNKNOWN");

    // L2 Cache Size
    err = hipDeviceGetAttribute(&value, hipDeviceAttributeL2CacheSize, 0);
    if (err == hipSuccess) {
        printf("L2 Cache Size: %d bytes (%.2f MB)\n", value, value / (1024.0 * 1024.0));
    }

    // Cache configuration
    hipFuncCache_t cacheConfig;
    err = hipDeviceGetCacheConfig(&cacheConfig);
    if (err == hipSuccess) {
        printf("Current cache preference: ");
        switch(cacheConfig) {
            case hipFuncCachePreferNone: printf("No preference\n"); break;
            case hipFuncCachePreferShared: printf("Prefer shared memory\n"); break;
            case hipFuncCachePreferL1: printf("Prefer L1 cache\n"); break;
            case hipFuncCachePreferEqual: printf("Equal L1/shared\n"); break;
            default: printf("Unknown\n"); break;
        }
    }

    // Memory transaction size (cache line related)
    printf("\nMemory Transaction Analysis:\n");
    printf("Expected cache line size: 128 bytes (32 floats)\n");
    printf("Your warp accesses: 32 consecutive floats\n");
    printf("Theoretical coalescing: PERFECT (1:1 ratio expected)\n");

    printf("=====================================\n\n");
}

// Comparison function for qsort (for percentile calculation)
static int compare_doubles(const void *a, const void *b) {
    double arg1 = *(const double*)a;
    double arg2 = *(const double*)b;

    if (arg1 < arg2) return -1;
    if (arg1 > arg2) return 1;
    return 0;
}

// Compute comprehensive statistics for an array of doubles
void compute_array_statistics(const double* array, int size, ArrayStats* stats) {
    if (size <= 0 || array == NULL || stats == NULL) {
        return;
    }

    // Copy array for sorting (needed for percentile)
    double* sorted_array = (double*)malloc(size * sizeof(double));
    memcpy(sorted_array, array, size * sizeof(double));

    // Calculate average and find min/max
    double sum = 0.0;
    stats->minimum = array[0];
    stats->maximum = array[0];

    for (int i = 0; i < size; i++) {
        sum += array[i];
        if (array[i] < stats->minimum) stats->minimum = array[i];
        if (array[i] > stats->maximum) stats->maximum = array[i];
    }
    stats->average = sum / size;

    // Calculate standard deviation
    double variance_sum = 0.0;
    for (int i = 0; i < size; i++) {
        double diff = array[i] - stats->average;
        variance_sum += diff * diff;
    }
    stats->std_dev = sqrt(variance_sum / size);

    // Sort array and calculate 95th percentile
    qsort(sorted_array, size, sizeof(double), compare_doubles);

    // Calculate 95th percentile index
    double percentile_index = 0.95 * (size - 1);
    int lower_index = (int)floor(percentile_index);
    int upper_index = (int)ceil(percentile_index);

    if (lower_index == upper_index) {
        stats->p95 = sorted_array[lower_index];
    } else {
        // Linear interpolation between the two closest values
        double weight = percentile_index - lower_index;
        stats->p95 = sorted_array[lower_index] * (1.0 - weight) + sorted_array[upper_index] * weight;
    }

    free(sorted_array);
}

// Unified kernel dispatch function that both benchmark and error analysis can use
// Function to map kernel name to KernelType
KernelType getKernelTypeFromName(const char* name) {
    if (strcmp(name, "naive") == 0) return KERNEL_NAIVE;
    if (strcmp(name, "tiled") == 0) return KERNEL_TILED;
    if (strcmp(name, "tiled_opt") == 0) return KERNEL_TILED_OPT;
    if (strcmp(name, "tiled_pairwise") == 0) return KERNEL_TILED_PAIRWISE;
    if (strcmp(name, "tiled_rect") == 0) return KERNEL_TILED_RECT;
    if (strcmp(name, "cublas") == 0) return KERNEL_CUBLAS;
    if (strcmp(name, "cublas_tensor") == 0) return KERNEL_CUBLAS_TENSOR;
    if (strcmp(name, "cutlass") == 0) return KERNEL_CUTLASS;
    if (strcmp(name, "cutlass_tensor") == 0) return KERNEL_CUTLASS_TENSOR;
    return static_cast<KernelType>(-1); // Return invalid value for unknown names
}

// Function to map matrix type name to MatrixType
MatrixType getMatrixTypeFromName(const char* name) {
    if (strcmp(name, "wellcond") == 0) return MATRIX_ODO_WELL_CONDITIONED;
    if (strcmp(name, "illcond") == 0) return MATRIX_ODO_ILL_CONDITIONED;
    if (strcmp(name, "zeromean") == 0) return MATRIX_ZEROMEAN;
    if (strcmp(name, "normal") == 0) return MATRIX_ZEROMEAN;  // Keep backward compatibility
    if (strcmp(name, "uniform") == 0) return MATRIX_UNIFORM;
    if (strcmp(name, "2powers") == 0) return MATRIX_SCALED_2POWERS;
    if (strcmp(name, "scaled") == 0) return MATRIX_SCALED_2POWERS;  // Keep backward compatibility
    if (strcmp(name, "rademacher") == 0) return MATRIX_RADEMACHER;
    if (strcmp(name, "skewed") == 0) return MATRIX_SKEW_MAGNITUDE;
    if (strcmp(name, "file") == 0) return MATRIX_FROM_FILE;
    return static_cast<MatrixType>(-1); // Return invalid value for unknown names
}

// Reverse conversion functions: enum to string
const char* kernelTypeToString(KernelType kernel_type) {
    switch(kernel_type) {
        case KERNEL_NAIVE: return "naive";
        case KERNEL_TILED: return "tiled";
        case KERNEL_TILED_OPT: return "tiled_opt";
        case KERNEL_TILED_PAIRWISE: return "tiled_pairwise";
        case KERNEL_TILED_RECT: return "tiled_rect";
        case KERNEL_CUBLAS: return "cublas";
        case KERNEL_CUBLAS_TENSOR: return "cublas_tensor";
        case KERNEL_CUTLASS: return "cutlass";
        case KERNEL_CUTLASS_TENSOR: return "cutlass_tensor";
        default: return "unknown";
    }
}

const char* matrixTypeToString(MatrixType matrix_type) {
    switch(matrix_type) {
        case MATRIX_ODO_WELL_CONDITIONED: return "wellcond";
        case MATRIX_ODO_ILL_CONDITIONED: return "illcond";
        case MATRIX_ZEROMEAN: return "zeromean";
        case MATRIX_UNIFORM: return "uniform";
        case MATRIX_SCALED_2POWERS: return "2powers";
        case MATRIX_RADEMACHER: return "rademacher";
        case MATRIX_SKEW_MAGNITUDE: return "skewed";
        case MATRIX_FROM_FILE: return "file";
        default: return "unknown";
    }
}

// Optimized kernel dispatch using function pointer table
typedef void (*KernelFunc)(float*, float*, float*, int, dim3, dim3);

static KernelFunc kernel_function_table[] = {
    launch_naive,           // KERNEL_NAIVE
    launch_tiled,           // KERNEL_TILED
    launch_tiled_opt,       // KERNEL_TILED_OPT
    launch_tiled_pairwise,  // KERNEL_TILED_PAIRWISE
    launch_tiled_rect,      // KERNEL_TILED_RECT
    launch_cublas,          // KERNEL_CUBLAS
    launch_cublas_tensor,   // KERNEL_CUBLAS_TENSOR
    launch_cutlass,         // KERNEL_CUTLASS
    launch_cutlass_tensor   // KERNEL_CUTLASS_TENSOR
};

void launch_kernel_by_type(KernelType kernel_type, float* d_A, float* d_B, float* d_C, int n, dim3 blocks, dim3 threads) {
    // Bounds check for safety
    if (kernel_type < 0 || kernel_type >= sizeof(kernel_function_table)/sizeof(kernel_function_table[0])) {
        printf("ERROR: Invalid kernel type %d\n", (int)kernel_type);
        return;
    }

    // Direct function pointer call - zero overhead dispatch!
    kernel_function_table[kernel_type](d_A, d_B, d_C, n, blocks, threads);
}
// main.cu
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include "benchmark.h"

void printUsage() {
    printf("Usage: ./main [options]\n\n");
    printf("Options:\n");
    printf("  --help           Show this help\n");
    printf("  --test=NAME      Run only specified test (naive, tiled, cublas)\n");
    printf("  --size=N         Run only specified matrix size\n");
    printf("  --all            Run all tests and sizes\n");
}

int main(int argc, char **argv) {
    // Add debug mode first
    if (argc > 1 && strcmp(argv[1], "--debug") == 0) {
        printf("=== Debug Mode ===\n");

        // Test basic CUDA
        int device_count;
        hipError_t err = hipGetDeviceCount(&device_count);
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
            return 1;
        }
        printf("✓ CUDA works, found %d device(s)\n", device_count);

        // Test device properties
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("✓ GPU: %s\n", prop.name);

        printf("✓ Debug test passed\n");
        return 0;
    }

    // Default: enable all tests and sizes
    bool enabled_tests[NUM_TESTS] = {true, true, true, true, true}; // All enabled by default
    bool enabled_sizes[NUM_SIZES] = {true, true, true, true, true}; // 5 elements: 256,512,1024,2048,4096

    // Parse command line arguments
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--help") == 0) {
            printUsage();
            return 0;
        } else if (strncmp(argv[i], "--test=", 7) == 0) {
            // Disable all tests first
            for (int j = 0; j < NUM_TESTS; j++) {
                enabled_tests[j] = false;
            }

            // Enable specific test
            const char* test_name = argv[i] + 7;
            bool found = false;

            for (int j = 0; j < NUM_TESTS; j++) {
                if (strcmp(available_tests[j].name, test_name) == 0) {
                    enabled_tests[j] = true;
                    found = true;
                    break;
                }
            }

            if (!found) {
                printf("Unknown test: %s\n", test_name);
                printf("Available tests: ");
                for (int j = 0; j < NUM_TESTS; j++) {
                    printf("%s ", available_tests[j].name);
                }
                printf("\n");
                return 1;
            }
        } else if (strncmp(argv[i], "--size=", 7) == 0) {
            // Disable all sizes first
            for (int j = 0; j < NUM_SIZES; j++) {
                enabled_sizes[j] = false;
            }

            // Enable specific size
            int target_size = atoi(argv[i] + 7);
            bool found = false;

            for (int j = 0; j < NUM_SIZES; j++) {
                if (SIZES[j] == target_size) {
                    enabled_sizes[j] = true;
                    found = true;
                    break;
                }
            }

            if (!found) {
                printf("Size %d not supported\n", target_size);
                printf("Available sizes: ");
                for (int j = 0; j < NUM_SIZES; j++) {
                    printf("%d ", SIZES[j]);
                }
                printf("\n");
                return 1;
            }
        } else if (strcmp(argv[i], "--all") == 0) {
            // Already enabled all by default
        } else {
            printf("Unknown option: %s\n", argv[i]);
            printUsage();
            return 1;
        }
    }

    printf("GEMM Performance Profiling\n");
    printf("==========================\n");

    // Initialize CUDA
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("GPU: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Memory: %.1f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));

    // Show which tests will run
    printf("\nEnabled tests: ");
    for (int i = 0; i < NUM_TESTS; i++) {
        if (enabled_tests[i]) {
            printf("%s ", available_tests[i].name);
        }
    }
    printf("\n");

    printf("Enabled sizes: ");
    for (int i = 0; i < NUM_SIZES; i++) {
        if (enabled_sizes[i]) {
            printf("%d ", SIZES[i]);
        }
    }
    printf("\n\n");

    // TEMPORARILY COMMENT OUT THE BENCHMARK CALL
    printf("About to call runAllBenchmarks...\n");
    fflush(stdout);

    // Start profiling if using CUDA profiler
    hipProfilerStart();

    // Run benchmarks - NOW PROPERLY IMPLEMENTED
    runAllBenchmarks(enabled_tests, enabled_sizes);

    // Stop profiling
    hipProfilerStop();

    printf("\nBenchmarking complete!\n");
    return 0;
}
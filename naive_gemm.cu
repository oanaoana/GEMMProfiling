#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matmul_naive(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k)
            sum += A[row * N + k] * B[k * N + col];
        C[row * N + col] = sum;
    }
}

void fill_matrix(float *mat, int N) {
    for (int i = 0; i < N*N; ++i)
        mat[i] = static_cast<float>(i % 100);
}

int main() {
    int N = 1024;
    size_t size = N * N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    fill_matrix(h_A, N);
    fill_matrix(h_B, N);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Warm-up run
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
    matmul_naive<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Timing run
    hipEventRecord(start);
    matmul_naive<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Calculate GFLOP/s
    // For matrix multiplication: 2*N^3 floating point operations
    double operations = 2.0 * N * N * N;  // 2*N^3 for N×N matrices
    double gigaFlops = (operations / (milliseconds / 1000.0)) / 1e9;

    printf("Matrix size: %d x %d\n", N, N);
    printf("Execution time: %f ms\n", milliseconds);
    printf("Performance: %f GFLOP/s\n", gigaFlops);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    printf("Result[0] = %f\n", h_C[0]);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    return 0;
}
